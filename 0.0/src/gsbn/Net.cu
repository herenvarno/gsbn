#include "hip/hip_runtime.h"
#include "gsbn/Net.hpp"
#include "gsbn/Database.hpp"

namespace gsbn{

__device__ int list_size_gpu;

void Net::update_phase_0_gpu(){
	int size = _rnd_uniform01->height();
	_rnd.gen_uniform01_gpu(static_cast<float *>(_rnd_uniform01->mutable_gpu_data()), _rnd_uniform01->height());
        
	int h_hcu = _hcu->height();
	int idx=0;
	for(int i=0; i<h_hcu; i++){
		int mcu_num = static_cast<const int *>(_hcu->cpu_data(i))[Database::IDX_HCU_MCU_NUM];
		float snoise = static_cast<const float *>(_hcu->cpu_data(i))[Database::IDX_HCU_SNOISE];
		_rnd.gen_normal_gpu(static_cast<float *>(_rnd_normal->mutable_gpu_data(idx)), mcu_num, 0, snoise);
		idx += mcu_num;
	}
}

/*
 * Phase 1: update DSUP
 */
__global__ void update_kernel_phase_1_gpu(
	int n,
	const void *ptr_mcu, int w_mcu,
	const void *ptr_addr, int w_addr,
	const void *ptr_hcu, int w_hcu,
	const void *ptr_stim, int w_stim,
	const void *ptr_rnd_normal, int w_rnd_normal,
	void *ptr_j_array, int w_j_array,
	void *ptr_epsc, int w_epsc,
	void *ptr_sup, int w_sup,
	float gain_mask){
	
	CUDA_KERNEL_LOOP(idx, n) {
		int hcu_idx = static_cast<const int*>(ptr_addr+idx*w_addr)[Database::IDX_ADDR_HCU];
		const int *ptr_hcu_data = static_cast<const int*>(ptr_hcu+hcu_idx*w_hcu);
		int hcu_isp_idx = ptr_hcu_data[Database::IDX_HCU_ISP_INDEX];
		int hcu_isp_num = ptr_hcu_data[Database::IDX_HCU_ISP_NUM];
		const float *ptr_hcu_data0 = static_cast<const float*>(ptr_hcu+hcu_idx*w_hcu);
		float taumdt = ptr_hcu_data0[Database::IDX_HCU_TAUMDT];
		float igain = ptr_hcu_data0[Database::IDX_HCU_IGAIN];
		float lgbias = ptr_hcu_data0[Database::IDX_HCU_LGBIAS];
		float wgain = ptr_hcu_data0[Database::IDX_HCU_WGAIN]*gain_mask;	// USE MASK
	
		float wsup=0;
		int j_array_idx=static_cast<const int*>(ptr_mcu+idx*w_mcu)[Database::IDX_MCU_J_ARRAY_INDEX];
	
		for(int i=0; i<hcu_isp_num; i++){
			float *ptr_j_array_data = static_cast<float *>(ptr_j_array+(j_array_idx+i)*w_j_array);
			float *ptr_epsc_data = static_cast<float *>(ptr_epsc+(j_array_idx+i)*w_epsc);
			float epsc = ptr_epsc_data[Database::IDX_EPSC_VALUE];
			float bj = ptr_j_array_data[Database::IDX_J_ARRAY_BJ];
			wsup += bj + epsc;
		}
	
		const float *ptr_stim_data=static_cast<const float *>(ptr_stim);
		const float *ptr_rnd_normal_data=static_cast<const float *>(ptr_rnd_normal+idx*w_rnd_normal);
		float sup = lgbias + igain * ptr_stim_data[idx] + ptr_rnd_normal_data[Database::IDX_RND_NORMAL_VALUE];
		sup += wgain * wsup;
	
		float *ptr_sup_data = static_cast<float *>(ptr_sup+idx*w_sup);
		float dsup=ptr_sup_data[Database::IDX_SUP_DSUP];
		ptr_sup_data[Database::IDX_SUP_DSUP] += (sup - dsup) * taumdt;
	}
}

void Net::update_phase_1_gpu(){

	const float *ptr_conf = static_cast<const float*>(_conf->cpu_data());
	float gain_mask=ptr_conf[Database::IDX_CONF_GAIN_MASK];
	int stim_idx= static_cast<const int*>(_conf->cpu_data())[Database::IDX_CONF_STIM];

	int h_mcu = _mcu->height();
	int w_mcu = _mcu->width();
	const void *ptr_mcu = _mcu->gpu_data();
	int w_addr = _addr->width();
	const void *ptr_addr = _addr->gpu_data();
	int w_hcu = _hcu->width();
	const void *ptr_hcu = _hcu->gpu_data();
	int w_stim = _stim->width();
	const void *ptr_stim = _stim->gpu_data(stim_idx);
	int w_rnd_normal = _rnd_normal->width();
	const void *ptr_rnd_normal = _rnd_normal->gpu_data();
	int w_j_array = _j_array->width();
	void *ptr_j_array = _j_array->mutable_gpu_data();
	int w_epsc = _epsc->width();
	void *ptr_epsc = _epsc->mutable_gpu_data();
	int w_sup = _sup->width();
	void *ptr_sup = _sup->mutable_gpu_data();
	
	update_kernel_phase_1_gpu<<<GSBN_GET_BLOCKS(h_mcu), GSBN_CUDA_NUM_THREADS>>>(
		h_mcu,
		ptr_mcu, w_mcu,
		ptr_addr, w_addr,
		ptr_hcu, w_hcu,
		ptr_stim, w_stim,
		ptr_rnd_normal, w_rnd_normal,
		ptr_j_array, w_j_array,
		ptr_epsc, w_epsc,
		ptr_sup, w_sup,
		gain_mask);
	CUDA_POST_KERNEL_CHECK;
}

/*
 * Phase 2: halfnormlize
 */
__global__ void update_kernel_phase_2_gpu(
	int n, 
	const void *ptr_hcu, int w_hcu,
	void *ptr_sup, int w_sup){
	
	CUDA_KERNEL_LOOP(idx, n) {
		const int *ptr_hcu_data = static_cast<const int *>(ptr_hcu+idx*w_hcu);
		int mcu_idx = ptr_hcu_data[Database::IDX_HCU_MCU_INDEX];
		int mcu_num = ptr_hcu_data[Database::IDX_HCU_MCU_NUM];
		const float *ptr_hcu_data0 = static_cast<const float *>(ptr_hcu+idx*w_hcu);
		float wtagain = ptr_hcu_data0[Database::IDX_HCU_WTAGAIN];
	
		float maxdsup = static_cast<float *>(ptr_sup+mcu_idx*w_sup)[Database::IDX_SUP_DSUP];
		for(int i=0; i<mcu_num; i++){
			float dsup=static_cast<float *>(ptr_sup+(mcu_idx+i)*w_sup)[Database::IDX_SUP_DSUP];
			if(dsup>maxdsup){
				maxdsup=dsup;
			}
		}
		float maxact = exp(wtagain*maxdsup);
		float vsum=0;
		for(int i=0; i<mcu_num; i++){
			float *ptr_sup_data = static_cast<float *>(ptr_sup+(mcu_idx+i)*w_sup);
			float dsup = ptr_sup_data[Database::IDX_SUP_DSUP];
			float act = exp(wtagain*(dsup-maxdsup));
			if(maxact<1){
				act *= maxact;
			}
			vsum += act;
			ptr_sup_data[Database::IDX_SUP_ACT]=act;
		}
	
		if(vsum>1){
			for(int i=0; i<mcu_num; i++){
				static_cast<float *>(ptr_sup+(mcu_idx+i)*w_sup)[Database::IDX_SUP_ACT]/=vsum;
			}
		}
	}
}
void Net::update_phase_2_gpu(){
	int h_hcu=_hcu->height();
	int w_hcu=_hcu->width();
	const void *ptr_hcu=_hcu->gpu_data();
	int w_sup=_sup->width();
	void *ptr_sup=_sup->mutable_gpu_data();
	
	update_kernel_phase_2_gpu<<<GSBN_GET_BLOCKS(h_hcu), GSBN_CUDA_NUM_THREADS>>>(
		h_hcu,
		ptr_hcu, w_hcu,
		ptr_sup, w_sup
	);
	CUDA_POST_KERNEL_CHECK;
}

/*
 * Phase 3: generate spike
 */
__global__ void update_kernel_phase_3_gpu(
	int n,
	const void *ptr_addr, int w_addr,
	const void *ptr_hcu, int w_hcu,
	const void *ptr_sup, int w_sup,
	const void *ptr_rnd_uniform01, int w_rnd_uniform01,
	void *ptr_spk, int w_spk){
	
	CUDA_KERNEL_LOOP(idx, n) {
		int hcu_idx = static_cast<const int*>(ptr_addr+idx*w_addr)[Database::IDX_ADDR_HCU];
		float maxfqdt = static_cast<const float*>(ptr_hcu+hcu_idx*w_hcu)[Database::IDX_HCU_MAXFQDT];
		unsigned char *ptr_spk_data = static_cast<unsigned char *>(ptr_spk+idx*w_spk);
		float act = static_cast<const float *>(ptr_sup+idx*w_sup)[Database::IDX_SUP_ACT];
		float randnum = static_cast<const float *>(ptr_rnd_uniform01+idx*w_rnd_uniform01)[Database::IDX_RND_UNIFORM01_VALUE];
		ptr_spk_data[Database::IDX_SPK_VALUE]=randnum < act*maxfqdt;
	}
}
void Net::update_phase_3_gpu(){
	int h_spk=_spk->height();
	int w_spk=_spk->width();
	void *ptr_spk=_spk->mutable_gpu_data();
	int w_addr = _addr->width();
	const void *ptr_addr = _addr->gpu_data();
	int w_hcu = _hcu->width();
	const void *ptr_hcu = _hcu->gpu_data();
	int w_sup=_sup->width();
	const void *ptr_sup=_sup->gpu_data(); 
	int w_rnd_uniform01=_rnd_uniform01->width();
	const void *ptr_rnd_uniform01=_rnd_uniform01->gpu_data();
	
	update_kernel_phase_3_gpu<<<GSBN_GET_BLOCKS(h_spk), GSBN_CUDA_NUM_THREADS>>>(
		h_spk,
		ptr_addr, w_addr,
		ptr_hcu, w_hcu,
		ptr_sup, w_sup,
		ptr_rnd_uniform01, w_rnd_uniform01,
		ptr_spk, w_spk);
	CUDA_POST_KERNEL_CHECK;
}

/*
 * Phase 4: generate short spike list, tmp table
 */

void Net::update_phase_4_gpu(){
	int h_spk=_spk->height();
	_tmp1->reset();
	for(int i=0; i<h_spk; i++){
		const unsigned char spike = static_cast<const unsigned char *>(_spk->cpu_data(i, 0))[0];
		if(spike){
			int *ptr_tmp1 = static_cast<int *>(_tmp1->expand(1));
			ptr_tmp1[0]=i;
		}
	}
}


/*
 * Phase 5: update EPSC, decrease
 */
__global__ void update_kernel_phase_5_gpu(
	int n,
	const void *ptr_mcu, int w_mcu,
	const void *ptr_addr, int w_addr,
	const void *ptr_hcu, int w_hcu,
	const void *ptr_hcu_isp, int w_hcu_isp,
	const void *ptr_proj, int w_proj,
	void *ptr_epsc, int w_epsc,
	float timestamp){
	
	CUDA_KERNEL_LOOP(idx, n) {
		int hcu_idx = static_cast<const int*>(ptr_addr+idx*w_addr)[Database::IDX_ADDR_HCU];
		const int *ptr_hcu_data = static_cast<const int*>(ptr_hcu+hcu_idx*w_hcu);
		int hcu_isp_idx = ptr_hcu_data[Database::IDX_HCU_ISP_INDEX];
		int hcu_isp_num = ptr_hcu_data[Database::IDX_HCU_ISP_NUM];
		int j_array_idx=static_cast<const int*>(ptr_mcu+idx*w_mcu)[Database::IDX_MCU_J_ARRAY_INDEX];
		for(int i=0; i<hcu_isp_num; i++){
			int proj_idx = static_cast<const int*>(ptr_hcu_isp+(hcu_isp_idx+i)*w_hcu_isp)[Database::IDX_HCU_ISP_VALUE];
			const float *ptr_proj_data = static_cast<const float*>(ptr_proj+proj_idx*w_proj);
			float kzi = ptr_proj_data[Database::IDX_PROJ_TAUZIDT];
			float *ptr_epsc_data = static_cast<float *>(ptr_epsc+(j_array_idx+i)*w_epsc);
			float epsc = ptr_epsc_data[Database::IDX_EPSC_VALUE];
			epsc *= (1-kzi);
			// FIXME
			//if(int(timestamp*10)%100==0 ){
			//	epsc = 0;
			//}
			ptr_epsc_data[Database::IDX_EPSC_VALUE] = epsc;
		}
	}
}
void Net::update_phase_5_gpu(){
	const float *ptr_conf = static_cast<const float*>(_conf->cpu_data());
	float dt=ptr_conf[Database::IDX_CONF_DT];
	float prn=ptr_conf[Database::IDX_CONF_PRN];
	float gain_mask=ptr_conf[Database::IDX_CONF_GAIN_MASK];
	int stim_idx= static_cast<const int*>(_conf->cpu_data())[Database::IDX_CONF_STIM];
	float timestamp=ptr_conf[Database::IDX_CONF_TIMESTAMP];

	int h_mcu = _mcu->height();
	int w_mcu = _mcu->width();
	const void *ptr_mcu = _mcu->gpu_data();
	int w_addr = _addr->width();
	const void *ptr_addr = _addr->gpu_data();
	int w_hcu = _hcu->width();
	const void *ptr_hcu = _hcu->gpu_data();
	int w_hcu_isp = _hcu_isp->width();
	const void *ptr_hcu_isp = _hcu_isp->gpu_data();
	int w_proj = _proj->width();
	const void *ptr_proj = _proj->gpu_data();
	int w_epsc = _epsc->width();
	void *ptr_epsc = _epsc->mutable_gpu_data();
	
	update_kernel_phase_5_gpu<<<GSBN_GET_BLOCKS(h_mcu), GSBN_CUDA_NUM_THREADS>>>(
		h_mcu,	// i is the index of "mcu" table
		ptr_mcu, w_mcu,
		ptr_addr, w_addr,
		ptr_hcu, w_hcu,
		ptr_hcu_isp, w_hcu_isp,
		ptr_proj, w_proj,
		ptr_epsc, w_epsc,
		timestamp);
}

/*
 * Phase 6: Scan and generate short coming spike list.
 */
void Net::update_phase_6_gpu(){
	int h_conn=_conn->height();
	_tmp2->reset();
	for(int i=0; i<h_conn; i++){
		int *ptr_conn = static_cast<int *>(_conn->mutable_cpu_data(i, 0));
		int queue=ptr_conn[Database::IDX_CONN_QUEUE];
		if(queue & 0x01){
			int *ptr_tmp2 = static_cast<int *>(_tmp2->expand(1));
			ptr_tmp2[Database::IDX_TMP2_CONN]=i;
			ptr_tmp2[Database::IDX_TMP2_DEST_HCU]=ptr_conn[Database::IDX_CONN_DEST_HCU];
			ptr_tmp2[Database::IDX_TMP2_SUBPROJ]=ptr_conn[Database::IDX_CONN_SUBPROJ];
			ptr_tmp2[Database::IDX_TMP2_PROJ]=ptr_conn[Database::IDX_CONN_PROJ];
			ptr_tmp2[Database::IDX_TMP2_IJ_MAT_INDEX]=ptr_conn[Database::IDX_CONN_IJ_MAT_INDEX];
		}
		ptr_conn[Database::IDX_CONN_QUEUE] = queue >> 1;
	}
}

/*
 * Phase 7: update IArray, IJMat, Wij, update EPSC, increase Zi2, increase Zi
 */
__global__ void update_kernel_phase_7_gpu(
	int n,
	const void *ptr_tmp2, int w_tmp2,
	const void *ptr_hcu, int w_hcu,
//	const void *ptr_hcu_isp, int w_hcu_isp,
	const void *ptr_proj, int w_proj,
	const void *ptr_mcu, int w_mcu,
	const void *ptr_j_array, int w_j_array,
	void *ptr_i_array, int w_i_array,
	void *ptr_ij_mat, int w_ij_mat,
	void *ptr_wij, int w_wij,
	void *ptr_epsc, int w_epsc,
	float timestamp, float prn, float gain_mask){
	
	CUDA_KERNEL_LOOP(idx, n) {
		const int *ptr_tmp2_data = static_cast<const int *>(ptr_tmp2+idx*w_tmp2);
		int idx_i_array = ptr_tmp2_data[Database::IDX_TMP2_CONN];
		int idx_ij_mat = ptr_tmp2_data[Database::IDX_TMP2_IJ_MAT_INDEX];
		int idx_wij = idx_ij_mat;
		int idx_hcu = ptr_tmp2_data[Database::IDX_TMP2_DEST_HCU];
		int idx_subproj = ptr_tmp2_data[Database::IDX_TMP2_SUBPROJ];
		int idx_proj = ptr_tmp2_data[Database::IDX_TMP2_PROJ];
	//	int idx_hcu_isp = static_cast<const int *>(ptr_hcu+idx_hcu*w_hcu)[Database::IDX_HCU_ISP_INDEX]+idx_subproj;
	//	int idx_proj = static_cast<const int *>(ptr_hcu_isp+idx_hcu_isp*w_hcu_isp)[Database::IDX_HCU_ISP_VALUE];
	
		const float *ptr_proj_data = static_cast<const float *>(ptr_proj+idx_proj*w_proj);
		float kp = ptr_proj_data[Database::IDX_PROJ_TAUPDT]*prn;
		float ke = ptr_proj_data[Database::IDX_PROJ_TAUEDT];
		float kzi = ptr_proj_data[Database::IDX_PROJ_TAUZIDT];
		float kzj = ptr_proj_data[Database::IDX_PROJ_TAUZJDT];
		//float wgain = ptr_proj_data[Database::IDX_PROJ_WGAIN]*gain_mask; //USE MASK ??? FIXME
		float wgain = ptr_proj_data[Database::IDX_PROJ_WGAIN]; //DONT USE MASK ???
		float eps = ptr_proj_data[Database::IDX_PROJ_EPS];
		float eps2 = ptr_proj_data[Database::IDX_PROJ_EPS2];
		float kfti = ptr_proj_data[Database::IDX_PROJ_KFTI];
		float prntaupdt = kp;
	
		float* ptr_i_array_data = static_cast<float *>(ptr_i_array+idx_i_array*w_i_array);
		float pi = ptr_i_array_data[Database::IDX_I_ARRAY_PI];
		float ei = ptr_i_array_data[Database::IDX_I_ARRAY_EI];
		float zi = ptr_i_array_data[Database::IDX_I_ARRAY_ZI];
		float ti = ptr_i_array_data[Database::IDX_I_ARRAY_TI];
		float pdt = timestamp - ti;
	
		int mcu_num = static_cast<const int *>(ptr_hcu+idx_hcu*w_hcu)[Database::IDX_HCU_MCU_NUM];
		for(int i=0; i<mcu_num; i++){
		
			float* ptr_ij_mat_data = static_cast<float *>(ptr_ij_mat+(idx_ij_mat+i)*w_ij_mat);
			float pij = ptr_ij_mat_data[Database::IDX_IJ_MAT_PIJ];
			float eij = ptr_ij_mat_data[Database::IDX_IJ_MAT_EIJ];
			float zi = ptr_ij_mat_data[Database::IDX_IJ_MAT_ZI2];
			float zj = ptr_ij_mat_data[Database::IDX_IJ_MAT_ZJ2];
			float tij = ptr_ij_mat_data[Database::IDX_IJ_MAT_TIJ];
			float pdt = timestamp - tij;
		
			//Update wij
			if(prntaupdt){
				int idx_mcu = static_cast<const int *>(ptr_hcu+idx_hcu*w_hcu)[Database::IDX_HCU_MCU_INDEX]+i;
				int idx_j_array = static_cast<const int *>(ptr_mcu+idx_mcu*w_mcu)[Database::IDX_MCU_J_ARRAY_INDEX]+idx_subproj;
				float pj = static_cast<const float *>(ptr_j_array+(idx_j_array)*w_j_array)[Database::IDX_J_ARRAY_PJ];		
				float* ptr_wij_data = static_cast<float *>(ptr_wij+(idx_wij+i)*w_wij);
				float wij = wgain * log((pij + eps2)/((pi + eps)*(pj + eps)));
				ptr_wij_data[Database::IDX_WIJ_VALUE] = wij;
				float *ptr_epsc_data = static_cast<float *>(ptr_epsc+(idx_j_array)*w_epsc);
				atomicAdd(&ptr_epsc_data[Database::IDX_EPSC_VALUE], wij);
			}
		
			//Update ij_mat
			if(pdt>0)
			{
				pij = (pij + ((eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi*zj)/(ke - kp) -
				            (ke*kp*zi*zj)/(kzi - kp + kzj))/(kzi - ke + kzj))/exp(kp*pdt) -
				    ((exp(kp*pdt - ke*pdt)*(eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi*zj))/(ke - kp) -
				     (ke*kp*zi*zj*exp(kp*pdt - kzi*pdt - kzj*pdt))/
				     (kzi - kp + kzj))/(exp(kp*pdt)*(kzi - ke + kzj));
				eij = (eij + (ke*zi*zj)/(kzi - ke + kzj))/exp(ke*pdt) -
				    (ke*zi*zj)/(exp(kzi*pdt)*exp(kzj*pdt)*(kzi - ke + kzj));
				zi = zi*exp(-kzi*pdt);
				zj = zj*exp(-kzj*pdt);
				tij = timestamp;
			 	
			 	ptr_ij_mat_data[Database::IDX_IJ_MAT_PIJ]=pij;
				ptr_ij_mat_data[Database::IDX_IJ_MAT_EIJ]=eij;
			
				ptr_ij_mat_data[Database::IDX_IJ_MAT_ZJ2]=zj;
				ptr_ij_mat_data[Database::IDX_IJ_MAT_TIJ]=tij;
			}
			// FIXME : is this correct?
			zi += kfti;
			ptr_ij_mat_data[Database::IDX_IJ_MAT_ZI2]=zi;
		

		}
	
		// update i_array
		if(pdt>0){
			pi = (pi - ((ei*kp*kzi - ei*ke*kp + ke*kp*zi)/(ke - kp) +
		                  (ke*kp*zi)/(kp - kzi))/(ke - kzi))/exp(kp*pdt) +
		      ((exp(kp*pdt - ke*pdt)*(ei*kp*kzi - ei*ke*kp + ke*kp*zi))/(ke - kp) +
		       (ke*kp*zi*exp(kp*pdt - kzi*pdt))/(kp - kzi))/(exp(kp*pdt)*(ke - kzi));
			ei = (ei - (ke*zi)/(ke - kzi))/exp(ke*pdt) +
		      (ke*zi*exp(ke*pdt - kzi*pdt))/(exp(ke*pdt)*(ke - kzi));
			zi = zi*exp(-kzi*pdt);
			ti = timestamp;
		
			ptr_i_array_data[Database::IDX_I_ARRAY_PI]=pi;
			ptr_i_array_data[Database::IDX_I_ARRAY_EI]=ei;
		
			ptr_i_array_data[Database::IDX_I_ARRAY_TI]=ti;
		}
		// FIXME : is this correct??
		zi += kfti;
		ptr_i_array_data[Database::IDX_I_ARRAY_ZI]=zi;
	}
}

void Net::update_phase_7_gpu(){
	int h_tmp2=_tmp2->height();
	if(h_tmp2<=0)
		return;
	const void *ptr_tmp2 = _tmp2->gpu_data();
	int w_tmp2 = _tmp2->width();
	void *ptr_wij = _wij -> mutable_gpu_data();
	int w_wij = _wij->width();
	void *ptr_i_array = _i_array -> mutable_gpu_data();
	int w_i_array = _i_array->width();
	const void *ptr_hcu = _hcu -> gpu_data();
	int w_hcu = _hcu->width();
	const void *ptr_proj = _proj -> gpu_data();
	int w_proj = _proj->width();
	const void *ptr_mcu = _mcu -> gpu_data();
	int w_mcu = _mcu->width();
	const void *ptr_j_array = _j_array -> gpu_data();
	int w_j_array = _j_array->width();
	void *ptr_ij_mat = _ij_mat -> mutable_gpu_data();
	int w_ij_mat = _ij_mat->width();
	void *ptr_epsc = _epsc -> mutable_gpu_data();
	int w_epsc = _epsc->width();
	
	const float *ptr_conf = static_cast<const float*>(_conf->cpu_data());
	float timestamp=ptr_conf[Database::IDX_CONF_TIMESTAMP];
	float prn=ptr_conf[Database::IDX_CONF_PRN];
	float gain_mask=ptr_conf[Database::IDX_CONF_GAIN_MASK];
	
	update_kernel_phase_7_gpu<<<GSBN_GET_BLOCKS(h_tmp2), GSBN_CUDA_NUM_THREADS>>>(
		h_tmp2,
		ptr_tmp2, w_tmp2,
		ptr_hcu, w_hcu,
		ptr_proj, w_proj,
		ptr_mcu, w_mcu,
		ptr_j_array, w_j_array,
		ptr_i_array, w_i_array,
		ptr_ij_mat, w_ij_mat,
		ptr_wij, w_wij,
		ptr_epsc, w_epsc,
		timestamp, prn, gain_mask);
	CUDA_POST_KERNEL_CHECK;
}

/*
 * Phase 8: update Pj, Ej, Zj
 * loop for #mcu times. inside each loop, it calculates #hcu_isp elements which
 * belongs to the same MCU.
 */
__global__ void update_kernel_phase_8_gpu(
	int n,
	const void *ptr_mcu, int w_mcu,
	const void *ptr_addr, int w_addr,
	const void *ptr_hcu, int w_hcu,
	const void *ptr_hcu_isp, int w_hcu_isp,
	const void *ptr_proj, int w_proj,
	void *ptr_j_array, int w_j_array,
	float pdt, float prn, float gain_mask){
	
	CUDA_KERNEL_LOOP(idx, n) {
		int hcu_idx = static_cast<const int*>(ptr_addr+idx*w_addr)[Database::IDX_ADDR_HCU];
		const int *ptr_hcu_data = static_cast<const int*>(ptr_hcu+hcu_idx*w_hcu);
		int hcu_isp_idx = ptr_hcu_data[Database::IDX_HCU_ISP_INDEX];
		int hcu_isp_num = ptr_hcu_data[Database::IDX_HCU_ISP_NUM];
		const float *ptr_hcu_data0 = static_cast<const float*>(ptr_hcu+hcu_idx*w_hcu);
		int j_array_idx=static_cast<const int*>(ptr_mcu+idx*w_mcu)[Database::IDX_MCU_J_ARRAY_INDEX];
		for(int i=0; i<hcu_isp_num; i++){
			int proj_idx = static_cast<const int*>(ptr_hcu_isp+(hcu_isp_idx+i)*w_hcu_isp)[Database::IDX_HCU_ISP_VALUE];
			const float *ptr_proj_data = static_cast<const float*>(ptr_proj+proj_idx*w_proj);
			float kzi = ptr_proj_data[Database::IDX_PROJ_TAUZIDT];
			float kzj = ptr_proj_data[Database::IDX_PROJ_TAUZJDT];
			float ke = ptr_proj_data[Database::IDX_PROJ_TAUEDT];
			float kp = ptr_proj_data[Database::IDX_PROJ_TAUPDT]*prn;
			float eps = ptr_proj_data[Database::IDX_PROJ_EPS];
			//float bgain = ptr_proj_data[Database::IDX_PROJ_BGAIN]*gain_mask; //USE MASK ??? FIXME
			float bgain = ptr_proj_data[Database::IDX_PROJ_BGAIN]; // DONT USE MASK ???
			float prntaupdt = kp;
		
			float *ptr_j_array_data = static_cast<float *>(ptr_j_array+(j_array_idx+i)*w_j_array);
			float pj = ptr_j_array_data[Database::IDX_J_ARRAY_PJ];
			float ej = ptr_j_array_data[Database::IDX_J_ARRAY_EJ];
			float zj = ptr_j_array_data[Database::IDX_J_ARRAY_ZJ];
			float bj;
		
			if(prntaupdt==0){
				bj = ptr_j_array_data[Database::IDX_J_ARRAY_BJ];
			}else{
				bj = bgain * log(pj + eps);
				ptr_j_array_data[Database::IDX_J_ARRAY_BJ]=bj;
			}
		
		
			pj = (pj - ((ej*kp*kzj - ej*ke*kp + ke*kp*zj)/(ke - kp) +
				                (ke*kp*zj)/(kp - kzj))/(ke - kzj))/exp(kp*pdt) +
				    ((exp(kp*pdt - ke*pdt)*(ej*kp*kzj - ej*ke*kp + ke*kp*zj))/(ke - kp) +
				     (ke*kp*zj*exp(kp*pdt - kzj*pdt))/(kp - kzj))/(exp(kp*pdt)*(ke - kzj));
			ej = (ej - (ke*zj)/(ke - kzj))/exp(ke*pdt) +
				    (ke*zj*exp(ke*pdt - kzj*pdt))/(exp(ke*pdt)*(ke - kzj));
			zj = zj*exp(-kzj*pdt);	
		
			ptr_j_array_data[Database::IDX_J_ARRAY_PJ] = pj;
			ptr_j_array_data[Database::IDX_J_ARRAY_EJ] = ej;
			ptr_j_array_data[Database::IDX_J_ARRAY_ZJ] = zj;
		}
	}
}
void Net::update_phase_8_gpu(){

	const float *ptr_conf = static_cast<const float*>(_conf->cpu_data());
	float dt=ptr_conf[Database::IDX_CONF_DT];
	float prn=ptr_conf[Database::IDX_CONF_PRN];
	float gain_mask=ptr_conf[Database::IDX_CONF_GAIN_MASK];
	int stim_idx= static_cast<const int*>(_conf->cpu_data())[Database::IDX_CONF_STIM];

	int h_mcu = _mcu->height();
	int w_mcu = _mcu->width();
	const void *ptr_mcu = _mcu->gpu_data();
	int w_addr = _addr->width();
	const void *ptr_addr = _addr->gpu_data();
	int w_hcu = _hcu->width();
	const void *ptr_hcu = _hcu->gpu_data();
	int w_hcu_isp = _hcu_isp->width();
	const void *ptr_hcu_isp = _hcu_isp->gpu_data();
	int w_proj = _proj->width();
	const void *ptr_proj = _proj->gpu_data();
	int w_j_array = _j_array->width();
	void *ptr_j_array = _j_array->mutable_gpu_data();
	
	update_kernel_phase_8_gpu<<<GSBN_GET_BLOCKS(h_mcu), GSBN_CUDA_NUM_THREADS>>>(
		h_mcu,
		ptr_mcu, w_mcu,
		ptr_addr, w_addr,
		ptr_hcu, w_hcu,
		ptr_hcu_isp, w_hcu_isp,
		ptr_proj, w_proj,
		ptr_j_array, w_j_array,
		dt, prn, gain_mask);
	CUDA_POST_KERNEL_CHECK;
}



/*
 * Phase 9: increase Zj
 * No need for timestamp
 */
__global__ void update_kernel_phase_9_gpu(
	int n, 
	const void *ptr_tmp1, int w_tmp1,
	const void *ptr_mcu, int w_mcu,
	const void *ptr_addr, int w_addr,
	const void *ptr_hcu, int w_hcu,
	const void *ptr_hcu_isp, int w_hcu_isp,
	const void *ptr_proj, int w_proj,
	void *ptr_j_array, int w_j_array){
	
	CUDA_KERNEL_LOOP(idx, n) {
		int mcu_idx = static_cast<const int *>(ptr_tmp1+idx*w_tmp1)[Database::IDX_TMP1_MCU_IDX];
		int hcu_idx = static_cast<const int *>(ptr_addr+mcu_idx*w_addr)[Database::IDX_ADDR_HCU];
		int hcu_isp_idx = static_cast<const int *>(ptr_hcu+hcu_idx*w_hcu)[Database::IDX_HCU_ISP_INDEX];
		const int *ptr_mcu_data = static_cast<const int *>(ptr_mcu+mcu_idx*w_mcu);
		int idx_j_array = ptr_mcu_data[Database::IDX_MCU_J_ARRAY_INDEX];
		int num_j_array = ptr_mcu_data[Database::IDX_MCU_J_ARRAY_NUM];
		for(int i=0; i<num_j_array; i++){
			int proj_idx = static_cast<const int *>(ptr_hcu_isp+(hcu_isp_idx+i)*w_hcu_isp)[Database::IDX_HCU_ISP_VALUE];
			float kftj = static_cast<const float *>(ptr_proj+proj_idx*w_proj)[Database::IDX_PROJ_KFTJ];
			float *ptr_j_array_data = static_cast<float *>(ptr_j_array+(idx_j_array+i)*w_j_array);
			ptr_j_array_data[Database::IDX_J_ARRAY_ZJ]+=kftj;
		}
	}
}
void Net::update_phase_9_gpu(){
	int h_tmp1=_tmp1->height();
	int w_tmp1=_tmp1->width();
	if(h_tmp1<=0)
		return;
	const void *ptr_tmp1=_tmp1->gpu_data();
	int w_mcu=_mcu->width();
	const void *ptr_mcu=_mcu->gpu_data();
	int w_addr = _addr->width();
	const void *ptr_addr = _addr->gpu_data();
	int w_hcu = _hcu->width();
	const void *ptr_hcu = _hcu->gpu_data();
	int w_hcu_isp = _hcu_isp->width();
	const void *ptr_hcu_isp = _hcu_isp->gpu_data();
	int w_proj = _proj->width();
	const void *ptr_proj = _proj->gpu_data();
	int w_j_array = _j_array->width();
	void * ptr_j_array = _j_array->mutable_gpu_data();
	
	update_kernel_phase_9_gpu<<<GSBN_GET_BLOCKS(h_tmp1), GSBN_CUDA_NUM_THREADS>>>(
		h_tmp1,
		ptr_tmp1, w_tmp1,
		ptr_mcu, w_mcu,
		ptr_addr, w_addr,
		ptr_hcu, w_hcu,
		ptr_hcu_isp, w_hcu_isp,
		ptr_proj, w_proj,
		ptr_j_array, w_j_array);
	CUDA_POST_KERNEL_CHECK;
}

/*
 * Phase 10: increase Zj2, update incoming spike
 */
__global__ void update_kernel_phase_10_gpu(
	int n,
	const void *ptr_tmp2, int w_tmp2,
	const void *ptr_proj, int w_proj,
	void *ptr_ij_mat, int w_ij_mat){
	
	CUDA_KERNEL_LOOP(idx, n) {
		const int *ptr_tmp2_data = static_cast<const int *>(ptr_tmp2+idx*w_tmp2);
		int idx_ij_mat = ptr_tmp2_data[Database::IDX_TMP2_IJ_MAT_INDEX];
		int idx_proj = ptr_tmp2_data[Database::IDX_TMP2_PROJ];
		const float *ptr_proj_data = static_cast<const float *>(ptr_proj+idx_proj*w_proj);
		float kftj = ptr_proj_data[Database::IDX_PROJ_KFTJ];
	
		float *ptr_ij_mat_data = static_cast<float *>(ptr_ij_mat+idx_ij_mat*w_ij_mat);
		ptr_ij_mat_data[Database::IDX_IJ_MAT_ZJ2] += kftj;
	}
}

void Net::update_phase_10_gpu(){
	_tmp2->reset();
	int h_tmp1=_tmp1->height();
	for(int i=0; i<h_tmp1; i++){
		int mcu=static_cast<const int *>(_tmp1->cpu_data(i))[Database::IDX_TMP1_MCU_IDX];
		int hcu=static_cast<const int *>(_addr->cpu_data(mcu))[Database::IDX_ADDR_HCU];
		int h_conn=_conn->height();
		for(int j=0; j<h_conn; j++){
			const int *ptr_conn = static_cast<const int *>(_conn->cpu_data(j));
			int src_mcu=ptr_conn[Database::IDX_CONN_SRC_MCU];
			int dest_hcu=ptr_conn[Database::IDX_CONN_DEST_HCU];
			if(dest_hcu==hcu){
				int ij_mat_first=static_cast<const int *>(_conn->cpu_data(j))[Database::IDX_CONN_IJ_MAT_INDEX];
				int offset = mcu - static_cast<const int *>(_hcu->cpu_data(hcu))[Database::IDX_HCU_MCU_INDEX];
				int ij_mat_idx = ij_mat_first + offset;
				int *ptr = static_cast<int*>(_tmp2->expand(1));
				ptr[Database::IDX_TMP2_IJ_MAT_INDEX]=ij_mat_idx;
				ptr[Database::IDX_TMP2_PROJ]=static_cast<const int *>(_conn->cpu_data(j))[Database::IDX_CONN_PROJ];
			}
			if(src_mcu==mcu){
				int delay = ptr_conn[Database::IDX_CONN_DELAY];
				static_cast<int *>(_conn->mutable_cpu_data(j))[Database::IDX_CONN_QUEUE] |= (1 << (delay-1));
			}
		}
	}
	int h_tmp2=_tmp2->height();
	if(h_tmp2<=0)
		return;
	const void *ptr_tmp2 = _tmp2->cpu_data();
	int w_tmp2 = _tmp2->width();
	const void *ptr_proj = _proj->cpu_data();
	int w_proj = _proj->width();
	void *ptr_ij_mat = _ij_mat->mutable_cpu_data();
	int w_ij_mat = _ij_mat->width();
	update_kernel_phase_10_gpu<<<GSBN_GET_BLOCKS(h_tmp2), GSBN_CUDA_NUM_THREADS>>>(
		h_tmp2,
		ptr_tmp2, w_tmp2,
		ptr_proj, w_proj,
		ptr_ij_mat, w_ij_mat);
	CUDA_POST_KERNEL_CHECK;
}


/*
 * Phase 11: deal with special spikes (REQ and ACK)
 */
void Net::update_phase_11_gpu(){
	_tmp3->reset();

	int plasticity = static_cast<const int *>(_conf->cpu_data(0))[Database::IDX_CONF_PLASTICITY];
	if(!plasticity)
		return;
	
	int h_conn0=_conn0->height();
	for(int i=0; i<h_conn0; i++){
		int *ptr_conn0 = static_cast<int *>(_conn0->mutable_cpu_data(i, 0));
		int queue=ptr_conn0[Database::IDX_CONN0_QUEUE];
		
		if(queue & 0x01){
			int *ptr_hcu_slot;
			int *ptr_conn;
			int *ptr_hcu;
			int idx_mcu_fanout;
			int *ptr_mcu_fanout;
			int idx_hcu;
			int mcu_num;
			void *ptr_tmp3;
			int* ptr_tmp30;
			float* ptr_tmp31;
			int proj_idx;
			int proj_mcu_num;
			float pi0;
			vector<int> *vec;
			vector<int>::iterator position;
			switch(ptr_conn0[Database::IDX_CONN0_TYPE]){
			case 1:	// REQ INCOMMING SPIKE
//				LOG(INFO) << ptr_conn0[Database::IDX_CONN0_DEST_HCU];
				ptr_hcu_slot = static_cast<int *>(_hcu_slot->mutable_cpu_data(ptr_conn0[Database::IDX_CONN0_DEST_HCU]));
				if(ptr_hcu_slot[Database::IDX_HCU_SLOT_VALUE]>0){
					ptr_hcu_slot[Database::IDX_HCU_SLOT_VALUE]--;
					ptr_conn0[Database::IDX_CONN0_TYPE]=2;
				}else{
					ptr_conn0[Database::IDX_CONN0_TYPE]=3;
				}
				queue |= (0x01 << (ptr_conn0[Database::IDX_CONN0_DELAY]));
				break;
			case 2:	// ACK INCOMMING SPIKE, ESTABLISH CONNECTION
				ptr_hcu = static_cast<int *>(_hcu->mutable_cpu_data(ptr_conn0[Database::IDX_CONN0_DEST_HCU]));
				mcu_num = ptr_hcu[Database::IDX_HCU_MCU_NUM];
				
				proj_idx = ptr_conn0[Database::IDX_CONN0_PROJ];
				pi0 = static_cast<const float *>(_proj->cpu_data(proj_idx))[Database::IDX_PROJ_PI0];
				proj_mcu_num = static_cast<const int *>(_proj->cpu_data(proj_idx))[Database::IDX_PROJ_MCU_NUM];
				// use tmp3 to initialize new connection
				ptr_tmp3 = _tmp3->expand(1);
				ptr_tmp30 = static_cast<int *>(ptr_tmp3);
				ptr_tmp30[Database::IDX_TMP3_CONN] = _conn->height();
				ptr_tmp30[Database::IDX_TMP3_DEST_HCU] = ptr_conn0[Database::IDX_CONN0_DEST_HCU];
				ptr_tmp30[Database::IDX_TMP3_IJ_MAT_IDX] = _ij_mat->height();
				ptr_tmp31 = static_cast<float *>(ptr_tmp3);
				ptr_tmp31[Database::IDX_TMP3_PI_INIT] = pi0;
				ptr_tmp31[Database::IDX_TMP3_PIJ_INIT] = 1.0/proj_mcu_num/mcu_num;
			
				ptr_conn = static_cast<int *>(_conn->expand(1));
				ptr_conn[Database::IDX_CONN_SRC_MCU] = ptr_conn0[Database::IDX_CONN0_SRC_MCU];
				ptr_conn[Database::IDX_CONN_DEST_HCU] = ptr_conn0[Database::IDX_CONN0_DEST_HCU];
				ptr_conn[Database::IDX_CONN_SUBPROJ] = ptr_conn0[Database::IDX_CONN0_SUBPROJ];
				ptr_conn[Database::IDX_CONN_PROJ] = ptr_conn0[Database::IDX_CONN0_PROJ];
				ptr_conn[Database::IDX_CONN_DELAY] = ptr_conn0[Database::IDX_CONN0_DELAY];
				ptr_conn[Database::IDX_CONN_QUEUE] = 0x01;
				ptr_conn[Database::IDX_CONN_IJ_MAT_INDEX] = _ij_mat->height();
				MemBlock::type_t t;
				_i_array->expand(1, &t);
				_ij_mat->expand(mcu_num, &t);
				_wij->expand(mcu_num, &t);
				ptr_conn0[Database::IDX_CONN0_TYPE] = 0;
				
				_empty_conn0_list.push_back(i);
				break;
			case 3:	// ACK INCOMMING SPIKE, REFUSE CONNECTION
				ptr_conn0[Database::IDX_CONN0_TYPE] = 0;	//set conn type to EMPTY, connection removed.
				idx_mcu_fanout = ptr_conn0[Database::IDX_CONN0_SRC_MCU];
				idx_hcu = ptr_conn0[Database::IDX_CONN0_DEST_HCU];
				ptr_mcu_fanout = static_cast<int*>(_mcu_fanout->mutable_cpu_data(idx_mcu_fanout));
				ptr_mcu_fanout[Database::IDX_MCU_FANOUT_VALUE]++;	// Recovery the fanout
				// Update the empty row list. It will be reused to establish new connections.
				_empty_conn0_list.push_back(i);
				vec = &(_existed_conn_list[idx_mcu_fanout]);
				position = find(vec->begin(), vec->end(), idx_hcu);
				if (position != vec->end())
					vec->erase(position);
				break;
			default:
				break;
			}
		}
		ptr_conn0[Database::IDX_CONN0_QUEUE] = queue >> 1;
	}
}


/*
 * Phase 12: initialize new IArray, IJMat
 */

__global__ void update_kernel_phase_12_gpu(
	int n,
	const void *ptr_tmp3, int w_tmp3,
	const void *ptr_hcu, int w_hcu,
	void *ptr_i_array, int w_i_array,
	void *ptr_ij_mat, int w_ij_mat,
	float timestamp){

	CUDA_KERNEL_LOOP(idx, n) {
		const int *ptr_tmp3_data=static_cast<const int*>(ptr_tmp3+idx*w_tmp3);
		int i_array_idx=ptr_tmp3_data[Database::IDX_TMP3_CONN];
		int hcu_idx=ptr_tmp3_data[Database::IDX_TMP3_DEST_HCU];
		int ij_mat_idx = ptr_tmp3_data[Database::IDX_TMP3_IJ_MAT_IDX];
		const float *ptr_tmp3_data0=static_cast<const float*>(ptr_tmp3+idx*w_tmp3);
		float pi_init = ptr_tmp3_data0[Database::IDX_TMP3_PI_INIT];
		float pij_init = ptr_tmp3_data0[Database::IDX_TMP3_PIJ_INIT];
		static_cast<float *>(ptr_i_array+i_array_idx*w_i_array)[Database::IDX_I_ARRAY_PI] = pi_init;
		static_cast<float *>(ptr_i_array+i_array_idx*w_i_array)[Database::IDX_I_ARRAY_TI] = timestamp;
		int mcu_num = static_cast<const int*>(ptr_hcu+hcu_idx*w_hcu)[Database::IDX_HCU_MCU_NUM];
		for(int i=0; i<mcu_num; i++){
			static_cast<float *>(ptr_ij_mat+(ij_mat_idx+i)*w_ij_mat)[Database::IDX_IJ_MAT_PIJ] = pij_init;
			static_cast<float *>(ptr_ij_mat+(ij_mat_idx+i)*w_ij_mat)[Database::IDX_IJ_MAT_TIJ] = timestamp;
		}
	}
}

void Net::update_phase_12_gpu(){
	int plasticity = static_cast<const int *>(_conf->cpu_data(0))[Database::IDX_CONF_PLASTICITY];
	if(!plasticity)
		return;
		
	int h_tmp3 = _tmp3->height();
	if(h_tmp3<=0){
		return;
	}
	const void *ptr_tmp3 = _tmp3->cpu_data();
	int w_tmp3 = _tmp3->width();
	const void *ptr_hcu = _hcu->cpu_data();
	int w_hcu = _hcu->width();
	void *ptr_i_array = _i_array->mutable_cpu_data();
	int w_i_array = _i_array->width();
	void *ptr_ij_mat = _ij_mat->mutable_cpu_data();
	int w_ij_mat = _ij_mat->width();
	
	float timestamp = *static_cast<const float*>(_conf->cpu_data(0, Database::IDX_CONF_TIMESTAMP));
	
	update_kernel_phase_12_gpu<<<GSBN_GET_BLOCKS(h_tmp3), GSBN_CUDA_NUM_THREADS>>>(
		h_tmp3,
		ptr_tmp3, w_tmp3,
		ptr_hcu, w_hcu,
		ptr_i_array, w_i_array,
		ptr_ij_mat, w_ij_mat,
		timestamp);
	CUDA_POST_KERNEL_CHECK;
}


/*
 * Phase 13: Send special spikes
 */
void Net::update_phase_13_gpu(){
	
	int plasticity = static_cast<const int *>(_conf->cpu_data(0))[Database::IDX_CONF_PLASTICITY];
	if(!plasticity)
		return;

	int h_tmp1 = _tmp1->height();
	for(int i=0; i<h_tmp1; i++){
		int idx_mcu = static_cast<const int *>(_tmp1->cpu_data(i))[Database::IDX_TMP1_MCU_IDX];
		int *ptr_mcu_fanout = static_cast<int *>(_mcu_fanout->mutable_cpu_data(idx_mcu));
		if(*ptr_mcu_fanout>0){
			*ptr_mcu_fanout--;
			
			const int *ptr_addr = static_cast<const int *>(_addr->cpu_data(idx_mcu));
			int idx_hcu=ptr_addr[Database::IDX_ADDR_HCU];

			const int *ptr_hcu = static_cast<const int *>(_hcu->cpu_data(idx_hcu));
			int idx_hcuproj=ptr_hcu[Database::IDX_HCU_OSP_INDEX];
			int num_hcuproj=ptr_hcu[Database::IDX_HCU_OSP_NUM];
			vector<int> proj_list;
			for(int j=0; j<num_hcuproj; j++){
				int proj_val = static_cast<const int *>(_hcu_osp->cpu_data(idx_hcuproj+j))[Database::IDX_HCU_OSP_VALUE];
				proj_list.push_back(proj_val);
			}
			vector<int> list_available_hcu;
			vector<int> list_available_proj;
			for(vector<int>::iterator it=proj_list.begin(); it!=proj_list.end(); it++){
				int dest_pop = static_cast<const int *>(_proj->cpu_data(*it))[Database::IDX_PROJ_DEST_POP];
				int iii_hcu = static_cast<const int *>(_pop->cpu_data(dest_pop))[Database::IDX_POP_HCU_INDEX];
				int nnn_hcu = static_cast<const int *>(_pop->cpu_data(dest_pop))[Database::IDX_POP_HCU_NUM];
				for(int k=0;k<nnn_hcu;k++){
					list_available_hcu.push_back(iii_hcu+k);
					list_available_proj.push_back(*it);
				}
			}
			vector<int> list=_existed_conn_list[idx_mcu];
			for(vector<int>::iterator it=list.begin(); it!=list.end();it++){
				vector<int>::iterator position = find(list_available_hcu.begin(), list_available_hcu.end(), *it);
				if (position != list_available_hcu.end()){
					list_available_hcu.erase(position);
					list_available_proj.erase(list_available_proj.begin()+distance(list_available_hcu.begin(), position));
				}
			}
			if(list_available_hcu.size()<=0){
				*ptr_mcu_fanout++;
				continue;
			}
			float random_number;
			_rnd.gen_uniform01_cpu(&random_number);
			int idx_target_hcu = ceil(random_number*list_available_hcu.size()-1);
			int target_hcu = list_available_hcu[idx_target_hcu];
			int target_proj = list_available_proj[idx_target_hcu];
			int target_subproj = 0;
			int target_hcu_isp_idx = static_cast<const int*>(_hcu->cpu_data(target_hcu))[Database::IDX_HCU_ISP_INDEX];
			int target_hcu_isp_num = static_cast<const int*>(_hcu->cpu_data(target_hcu))[Database::IDX_HCU_ISP_NUM];
			for(int l=0;l<target_hcu_isp_num;l++){
				if(static_cast<const int*>(_hcu_isp->cpu_data(target_hcu_isp_idx+l))[Database::IDX_HCU_ISP_VALUE] == target_proj){
					target_subproj=l;
					break;
				}
			}
			
			int *ptr_conn0;
			if(_empty_conn0_list.empty()){
				ptr_conn0 = static_cast<int*>(_conn0->expand(1));
			}else{
				int index = _empty_conn0_list.back();
				_empty_conn0_list.pop_back();
				ptr_conn0 = static_cast<int*>(_conn0->mutable_cpu_data(index));
			}
			
			ptr_conn0[Database::IDX_CONN0_SRC_MCU] = idx_mcu;
			ptr_conn0[Database::IDX_CONN0_DEST_HCU] = target_hcu;
			ptr_conn0[Database::IDX_CONN0_SUBPROJ] = target_subproj;
			ptr_conn0[Database::IDX_CONN0_PROJ] = target_proj;
			ptr_conn0[Database::IDX_CONN0_DELAY] = __DELAY__;	// FIXME
			ptr_conn0[Database::IDX_CONN0_QUEUE] = 1 << __DELAY__-1; // FIXME
			ptr_conn0[Database::IDX_CONN0_TYPE] = 1;
			
			_existed_conn_list[idx_mcu].push_back(target_hcu);
		}
	}
}

}
