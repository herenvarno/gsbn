#include "hip/hip_runtime.h"
#include "gsbn/procedures/ProcFix/Proj.hpp"

#ifndef CPU_ONLY

#include "gsbn/Fix16.cuh"

namespace gsbn{
namespace proc_fix{

__global__ void update_full_kernel_gpu(
	int dim_conn,
	int dim_mcu,
	fix16 *ptr_pi,
	fix16 *ptr_ei,
	fix16 *ptr_zi,
	int *ptr_ti,
	const fix16 *ptr_pj,
	fix16 *ptr_pij,
	fix16 *ptr_eij,
	fix16 *ptr_zi2,
	fix16 *ptr_zj2,
	int *ptr_tij,
	fix16 *ptr_wij,
	int simstep,
	float kp,
	float ke,
	float kzi,
	float kzj,
	float wgain,
	float eps,
	float eps2,
	int norm_frac_bit,
	int p_frac_bit
){
	int i=blockIdx.y*gridDim.x+blockIdx.x;
	int j=threadIdx.x;
	
	__shared__ float sh_pi;
	if(j==0){
		float pi = fix16_to_fp32_gpu(ptr_pi[i], p_frac_bit);
		float zi = fix16_to_fp32_gpu(ptr_zi[i], norm_frac_bit);
		int ti = ptr_ti[i];
		int pdt = simstep - ti;
		if(pdt<=0){
			ptr_ti[i]=simstep;
		}else{
			float ei = fix16_to_fp32_gpu(ptr_ei[i], norm_frac_bit);
			pi = (pi - ((ei*kp*kzi - ei*ke*kp + ke*kp*zi)/(ke - kp) +
				(ke*kp*zi)/(kp - kzi))/(ke - kzi))/exp(kp*pdt) +
				((exp(kp*pdt - ke*pdt)*(ei*kp*kzi - ei*ke*kp + ke*kp*zi))/(ke - kp) +
				(ke*kp*zi*exp(kp*pdt - kzi*pdt))/(kp - kzi))/(exp(kp*pdt)*(ke - kzi));
			ei = (ei - (ke*zi)/(ke - kzi))/exp(ke*pdt) +
				(ke*zi*exp(ke*pdt - kzi*pdt))/(exp(ke*pdt)*(ke - kzi));
			zi = zi*exp(-kzi*pdt);
			ti = simstep;
		
			ptr_pi[i] = fp32_to_fix16_gpu(pi, p_frac_bit);
			ptr_ei[i] = fp32_to_fix16_gpu(ei, norm_frac_bit);
			ptr_zi[i] = fp32_to_fix16_gpu(zi, norm_frac_bit);
			ptr_ti[i] = ti;
		}
		sh_pi = pi;
	}
	__syncthreads();
	
	int index = i*dim_mcu+j;
	
	int tij = ptr_tij[index];
	float zi2 = fix16_to_fp32_gpu(ptr_zi2[index], norm_frac_bit);
	int pdt = simstep - tij;
	if(pdt<=0){
		ptr_tij[index]=simstep;
	}else{
		float pij = fix16_to_fp32_gpu(ptr_pij[index], p_frac_bit);
		float eij = fix16_to_fp32_gpu(ptr_eij[index], norm_frac_bit);
		float zj2 = fix16_to_fp32_gpu(ptr_zj2[index], norm_frac_bit);
	
		pij = (pij + ((eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2)/(ke - kp) -
			(ke*kp*zi2*zj2)/(kzi - kp + kzj))/(kzi - ke + kzj))/exp(kp*pdt) -
			((exp(kp*pdt - ke*pdt)*(eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2))/(ke - kp) -
			(ke*kp*zi2*zj2*exp(kp*pdt - kzi*pdt - kzj*pdt))/
			(kzi - kp + kzj))/(exp(kp*pdt)*(kzi - ke + kzj));
		eij = (eij + (ke*zi2*zj2)/(kzi - ke + kzj))/exp(ke*pdt) -
			(ke*zi2*zj2)/(exp(kzi*pdt)*exp(kzj*pdt)*(kzi - ke + kzj));
		zi2 = zi2*exp(-kzi*pdt);
		zj2 = zj2*exp(-kzj*pdt);
		tij = simstep;
			 	
		ptr_pij[index] = fp32_to_fix16_gpu(pij, p_frac_bit);
		ptr_eij[index] = fp32_to_fix16_gpu(eij, norm_frac_bit);
		ptr_zi2[index] = fp32_to_fix16_gpu(zi2, norm_frac_bit);
		ptr_zj2[index] = fp32_to_fix16_gpu(zj2, norm_frac_bit);
		ptr_tij[index] = tij;
			
		// update wij and epsc
		float wij;
		if(kp){
			float pi = sh_pi;
			float pj = fix16_to_fp32_gpu(ptr_pj[i/dim_conn*dim_mcu + j], p_frac_bit);
			wij = wgain * log((pij + eps2)/((pi + eps)*(pj + eps)));
			ptr_wij[index] = fp32_to_fix16_gpu(wij, norm_frac_bit);
		}
	}
}

__global__ void update_j_kernel_gpu(
	int n,
	const int8_t *ptr_sj,
	fix16 *ptr_pj,
	fix16 *ptr_ej,
	fix16 *ptr_zj,
	fix16 *ptr_bj,
	fix16 *ptr_epsc,
	float kp,
	float ke,
	float kzj,
	float kzi,
	float kftj,
	float bgain,
	float eps,
	int norm_frac_bit,
	int p_frac_bit
){
	CUDA_KERNEL_LOOP(idx, n){
		float pj = fix16_to_fp32_gpu(ptr_pj[idx], p_frac_bit);
		float ej = fix16_to_fp32_gpu(ptr_ej[idx], norm_frac_bit);
		float zj = fix16_to_fp32_gpu(ptr_zj[idx], norm_frac_bit);
		int8_t sj = ptr_sj[idx];

		float epsc = fix16_to_fp32_gpu(ptr_epsc[idx], norm_frac_bit);
		ptr_epsc[idx] = fp32_to_fix16_gpu(epsc*(1-kzi), norm_frac_bit);

		if(kp){
			float bj = bgain * log(pj + eps);
			ptr_bj[idx]=fp32_to_fix16_gpu(bj, norm_frac_bit);
		}

		pj += (ej - pj)*kp;
		ej += (zj - ej)*ke;
		zj *= (1-kzj);
		if(sj>0){
			zj += kftj;
		}
	
		ptr_pj[idx] = fp32_to_fix16_gpu(pj, p_frac_bit);
		ptr_ej[idx] = fp32_to_fix16_gpu(ej, norm_frac_bit);
		ptr_zj[idx] = fp32_to_fix16_gpu(zj, norm_frac_bit);
	}
}

__global__ void update_row_kernel_gpu(
	int dim_conn,
	int dim_mcu,
	const int *ptr_ssi,
	fix16 *ptr_pi,
	fix16 *ptr_ei,
	fix16 *ptr_zi,
	int *ptr_ti,
	const fix16 *ptr_pj,
	fix16 *ptr_pij,
	fix16 *ptr_eij,
	fix16 *ptr_zi2,
	fix16 *ptr_zj2,
	int *ptr_tij,
	fix16* ptr_wij,
	fix16* ptr_epsc,
	int simstep,
	float kp,
	float ke,
	float kzi,
	float kzj,
	float kfti,
	float wgain,
	float eps,
	float eps2,
	int norm_frac_bit,
	int p_frac_bit
){

	int i = blockIdx.x;
	int j = threadIdx.x;
	int row = ptr_ssi[i];
	int col = j;
	int index = row*dim_mcu+col;
	
	__shared__ float sh_pi;
	
	if(j==0){
		float pi = fix16_to_fp32_gpu(ptr_pi[row], p_frac_bit);
		float zi = fix16_to_fp32_gpu(ptr_zi[row], norm_frac_bit);
		int ti = ptr_ti[row];
		int pdt = simstep - ti;
		if(pdt<=0){
			zi += kfti;
			ptr_zi[row] = fp32_to_fix16_gpu(zi, norm_frac_bit);
			ptr_ti[row] = simstep;
		}else{
			float ei = fp32_to_fix16_gpu(ptr_ei[row], norm_frac_bit);
		
			pi = (pi - ((ei*kp*kzi - ei*ke*kp + ke*kp*zi)/(ke - kp) +
				(ke*kp*zi)/(kp - kzi))/(ke - kzi))/exp(kp*pdt) +
				((exp(kp*pdt - ke*pdt)*(ei*kp*kzi - ei*ke*kp + ke*kp*zi))/(ke - kp) +
				(ke*kp*zi*exp(kp*pdt - kzi*pdt))/(kp - kzi))/(exp(kp*pdt)*(ke - kzi));
			ei = (ei - (ke*zi)/(ke - kzi))/exp(ke*pdt) +
				(ke*zi*exp(ke*pdt - kzi*pdt))/(exp(ke*pdt)*(ke - kzi));
			zi = zi*exp(-kzi*pdt) + kfti;
			ti = simstep;
			ptr_pi[row] = fp32_to_fix16_gpu(pi, p_frac_bit);
			ptr_ei[row] = fp32_to_fix16_gpu(ei, norm_frac_bit);
			ptr_zi[row] = fp32_to_fix16_gpu(zi, norm_frac_bit);
			ptr_ti[row] = ti;
		}
		sh_pi = pi;
	}
	
	__syncthreads();
	float pij = fix16_to_fp32_gpu(ptr_pij[index], p_frac_bit);
	int tij = ptr_tij[index];
	float zi2 = fix16_to_fp32_gpu(ptr_zi2[index], norm_frac_bit);
	int pdt = simstep - tij;
	if(pdt<=0){
		zi2 += kfti;
		ptr_zi2[index] = fp32_to_fix16_gpu(zi2, norm_frac_bit);
		ptr_tij[index] = simstep;
	}else{
		float eij = fix16_to_fp32_gpu(ptr_eij[index], norm_frac_bit);
		float zj2 = fix16_to_fp32_gpu(ptr_zj2[index], norm_frac_bit);
	
		pij = (pij + ((eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2)/(ke - kp) -
			(ke*kp*zi2*zj2)/(kzi - kp + kzj))/(kzi - ke + kzj))/exp(kp*pdt) -
			((exp(kp*pdt - ke*pdt)*(eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2))/(ke - kp) -
			(ke*kp*zi2*zj2*exp(kp*pdt - kzi*pdt - kzj*pdt))/
			(kzi - kp + kzj))/(exp(kp*pdt)*(kzi - ke + kzj));
		eij = (eij + (ke*zi2*zj2)/(kzi - ke + kzj))/exp(ke*pdt) -
			(ke*zi2*zj2)/(exp(kzi*pdt)*exp(kzj*pdt)*(kzi - ke + kzj));
		zi2 = zi2*exp(-kzi*pdt)+kfti;
		zj2 = zj2*exp(-kzj*pdt);
		tij = simstep;
			 	
		ptr_pij[index] = fp32_to_fix16_gpu(pij, p_frac_bit);
		ptr_eij[index] = fp32_to_fix16_gpu(eij, norm_frac_bit);
		ptr_zi2[index] = fp32_to_fix16_gpu(zi2, norm_frac_bit);
		ptr_zj2[index] = fp32_to_fix16_gpu(zj2, norm_frac_bit);
		ptr_tij[index] = tij;
		
		float wij;
		int idx_hcu = row / dim_conn;
		int idx_mcu = idx_hcu * dim_mcu + j;
		if(kp){
			float pi = sh_pi;
			float pj = fix16_to_fp32_gpu(ptr_pj[idx_mcu], p_frac_bit);
			wij = wgain * log((pij + eps2)/((pi + eps)*(pj + eps)));
			ptr_wij[index] = fp32_to_fix16_gpu(wij, norm_frac_bit);
		}else{
			wij = fix16_to_fp32_gpu(ptr_wij[index], norm_frac_bit);
		}
		atomic_add_fp32_to_fix16_gpu(&ptr_epsc[idx_mcu], wij, norm_frac_bit);
	}
}

__global__ void update_col_kernel_gpu(
	int dim_conn,
	int dim_mcu,
	const int *ptr_ii,
	const int *ptr_ssj,
	fix16 *ptr_pij,
	fix16 *ptr_eij,
	fix16 *ptr_zi2,
	fix16 *ptr_zj2,
	int *ptr_tij,
	int simstep,
	float kp,
	float ke,
	float kzi,
	float kzj,
	float kftj,
	int norm_frac_bit,
	int p_frac_bit
){

	int i = blockIdx.x;
	int j = threadIdx.x;

	int row = ptr_ssj[j]/dim_mcu*dim_conn+i;
	if(ptr_ii[row]<0){
		return;
	}
	int col = ptr_ssj[j]%dim_mcu;
	int index = row*dim_mcu+col;
	
	int tij = ptr_tij[index];
	float zj2 = fix16_to_fp32_gpu(ptr_zj2[index], norm_frac_bit);
	int pdt = simstep - tij;
	if(pdt<=0){
		zj2 += kftj;
		ptr_zj2[index]= fp32_to_fix16_gpu(zj2, norm_frac_bit);
		ptr_tij[index]=simstep;
	}else{
		float pij = fix16_to_fp32_gpu(ptr_pij[index], p_frac_bit);
		float eij = fix16_to_fp32_gpu(ptr_eij[index], norm_frac_bit);
		float zi2 = fix16_to_fp32_gpu(ptr_zi2[index], norm_frac_bit);
	
		pij = (pij + ((eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2)/(ke - kp) -
			(ke*kp*zi2*zj2)/(kzi - kp + kzj))/(kzi - ke + kzj))/exp(kp*pdt) -
			((exp(kp*pdt - ke*pdt)*(eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2))/(ke - kp) -
			(ke*kp*zi2*zj2*exp(kp*pdt - kzi*pdt - kzj*pdt))/
			(kzi - kp + kzj))/(exp(kp*pdt)*(kzi - ke + kzj));
		eij = (eij + (ke*zi2*zj2)/(kzi - ke + kzj))/exp(ke*pdt) -
			(ke*zi2*zj2)/(exp(kzi*pdt)*exp(kzj*pdt)*(kzi - ke + kzj));
		zi2 = zi2*exp(-kzi*pdt);
		zj2 = zj2*exp(-kzj*pdt)+kftj;
		tij = simstep;
			 	
		ptr_pij[index] = fp32_to_fix16_gpu(pij, p_frac_bit);
		ptr_eij[index] = fp32_to_fix16_gpu(eij, norm_frac_bit);
		ptr_zi2[index] = fp32_to_fix16_gpu(zi2, norm_frac_bit);
		ptr_zj2[index] = fp32_to_fix16_gpu(zj2, norm_frac_bit);
		ptr_tij[index] = tij;
	}
}


void Proj::update_full_gpu(){
	const int *ptr_conf0 = static_cast<const int*>(_conf->cpu_data());
	const float *ptr_conf1 = static_cast<const float*>(_conf->cpu_data());
	int simstep = ptr_conf0[Database::IDX_CONF_TIMESTAMP];
	float prn = ptr_conf1[Database::IDX_CONF_PRN];
	float old_prn = ptr_conf1[Database::IDX_CONF_OLD_PRN];
	if(old_prn!=prn){
		fix16 *ptr_pi = _pi->mutable_gpu_data();
		fix16 *ptr_ei = _ei->mutable_gpu_data();
		fix16 *ptr_zi = _zi->mutable_gpu_data();
		int *ptr_ti = _ti->mutable_gpu_data();
		const fix16 *ptr_pj = _pj->gpu_data();
		fix16 *ptr_pij = _pij->mutable_gpu_data();
		fix16 *ptr_eij = _eij->mutable_gpu_data();
		fix16 *ptr_zi2 = _zi2->mutable_gpu_data();
		fix16 *ptr_zj2 = _zj2->mutable_gpu_data();
		int *ptr_tij = _tij->mutable_gpu_data();
		fix16 *ptr_wij = _wij->mutable_gpu_data();
		const dim3 GRID_SIZE(_dim_conn, _dim_hcu);
		update_full_kernel_gpu<<<GRID_SIZE, _dim_mcu, 0, _stream>>>(
			_dim_conn,
			_dim_mcu,
			ptr_pi,
			ptr_ei,
			ptr_zi,
			ptr_ti,
			ptr_pj,
			ptr_pij,
			ptr_eij,
			ptr_zi2,
			ptr_zj2,
			ptr_tij,
			ptr_wij,
			simstep,
			_taupdt*old_prn,
			_tauedt,
			_tauzidt,
			_tauzjdt,
			_wgain,
			_eps,
			_eps2,
			_norm_frac_bit,
			_p_frac_bit
		);
		CUDA_POST_KERNEL_CHECK;
	}
}

void Proj::update_j_gpu(){
	const float *ptr_conf = static_cast<const float*>(_conf->cpu_data());
	float prn = ptr_conf[Database::IDX_CONF_PRN];
	fix16 *ptr_pj = _pj->mutable_gpu_data();
	fix16 *ptr_ej = _ej->mutable_gpu_data();
	fix16 *ptr_zj = _zj->mutable_gpu_data();
	fix16 *ptr_bj = _bj->mutable_gpu_data()+_proj_in_pop*_dim_hcu*_dim_mcu;
	fix16 *ptr_epsc = _epsc->mutable_gpu_data()+_proj_in_pop*_dim_hcu*_dim_mcu;
	const int8_t *ptr_sj = _sj->gpu_data();

	update_j_kernel_gpu<<<GSBN_GET_BLOCKS(_dim_hcu*_dim_mcu), GSBN_GET_THREADS(_dim_hcu*_dim_mcu), 0, _stream>>>(
		_dim_hcu*_dim_mcu,
		ptr_sj,
		ptr_pj,
		ptr_ej,
		ptr_zj,
		ptr_bj,
		ptr_epsc,
		_taupdt*prn,
		_tauedt,
		_tauzjdt,
		_tauzidt,
		_kftj,
		_bgain,
		_eps,
		_norm_frac_bit,
		_p_frac_bit
	);
	CUDA_POST_KERNEL_CHECK;
}

void Proj::update_ss_gpu(){
	// get active in spike
	CONST_HOST_VECTOR(int8_t, *v_si) = _si->cpu_vector();
	CONST_HOST_VECTOR(int, *v_ii) = _ii->cpu_vector();
	CONST_HOST_VECTOR(int, *v_di) = _di->cpu_vector();
	HOST_VECTOR(int, *v_qi) = _qi->mutable_cpu_vector();
	HOST_VECTOR(int, *v_ssi) = _ssi->mutable_cpu_vector();
	
        v_ssi->clear();
        for(int i=0; i<_dim_conn * _dim_hcu; i++){
                if((*v_ii)[i]<0){
                        continue;
                }
                (*v_qi)[i] >>= 1;
                if((*v_qi)[i] & 0x01){
                        v_ssi->push_back(i);
                }

                int spk = (*v_si)[(*v_ii)[i]];
                if(spk){
                        (*v_qi)[i] |= (0x01 << (*v_di)[i]);
                }
        }

        // get active out spike
        CONST_HOST_VECTOR(int8_t, *v_sj) = _sj->cpu_vector();
        HOST_VECTOR(int, *v_ssj) = _ssj->mutable_cpu_vector();
        v_ssj->clear();
        for(int i=0; i<_dim_hcu * _dim_mcu; i++){
                if((*v_sj)[i]>0){
                        v_ssj->push_back(i);
                }
        }

}

void Proj::update_row_gpu(){
	int active_row_num = _ssi->gpu_vector()->size();
	if(active_row_num<=0){
		return;
	}

	const int *ptr_conf0 = static_cast<const int*>(_conf->cpu_data());
	const float *ptr_conf1 = static_cast<const float*>(_conf->cpu_data());
	int simstep = ptr_conf0[Database::IDX_CONF_TIMESTAMP];
	float prn = ptr_conf1[Database::IDX_CONF_PRN];
	
	fix16 *ptr_pi = _pi->mutable_gpu_data();
	fix16 *ptr_ei = _ei->mutable_gpu_data();
	fix16 *ptr_zi = _zi->mutable_gpu_data();
	int *ptr_ti = _ti->mutable_gpu_data();
	const fix16 *ptr_pj = _pj->gpu_data();
	fix16 *ptr_pij = _pij->mutable_gpu_data();
	fix16 *ptr_eij = _eij->mutable_gpu_data();
	fix16 *ptr_zi2 = _zi2->mutable_gpu_data();
	fix16 *ptr_zj2 = _zj2->mutable_gpu_data();
	int *ptr_tij = _tij->mutable_gpu_data();
	fix16 *ptr_wij = _wij->mutable_gpu_data();
	fix16 *ptr_epsc = _epsc->mutable_gpu_data()+ _proj_in_pop * _dim_hcu * _dim_mcu;
	
	const int *ptr_ssi = _ssi->gpu_data();

	update_row_kernel_gpu<<<active_row_num, _dim_mcu, 0, _stream>>>(
		_dim_conn,
		_dim_mcu,
		ptr_ssi,
		ptr_pi,
		ptr_ei,
		ptr_zi,
		ptr_ti,
		ptr_pj,
		ptr_pij,
		ptr_eij,
		ptr_zi2,
		ptr_zj2,
		ptr_tij,
		ptr_wij,
		ptr_epsc,
		simstep,
		_taupdt*prn,
		_tauedt,
		_tauzidt,
		_tauzjdt,
		_kfti,
		_wgain,
		_eps,
		_eps2,
		_norm_frac_bit,
		_p_frac_bit
	);
	CUDA_POST_KERNEL_CHECK;
	
	ptr_wij= _wij->mutable_cpu_data();
	for(int i=0; i<100; i++){
		cout << fix16_to_fp32(ptr_wij[i], _norm_frac_bit) << ",";
	}
	cout << endl;
}

void Proj::update_col_gpu(){
	int active_col_num = _ssj->gpu_vector()->size();
	if(active_col_num<=0){
		return;
	}
	const int *ptr_conf0 = static_cast<const int*>(_conf->cpu_data());
	const float *ptr_conf1 = static_cast<const float*>(_conf->cpu_data());
	int simstep = ptr_conf0[Database::IDX_CONF_TIMESTAMP];
	float prn = ptr_conf1[Database::IDX_CONF_PRN];
	
	fix16 *ptr_pij = _pij->mutable_gpu_data();
	fix16 *ptr_eij = _eij->mutable_gpu_data();
	fix16 *ptr_zi2 = _zi2->mutable_gpu_data();
	fix16 *ptr_zj2 = _zj2->mutable_gpu_data();
	int *ptr_tij = _tij->mutable_gpu_data();
	
	const int *ptr_ii = _ii->gpu_data();
	const int *ptr_ssj = _ssj->gpu_data();
	
	update_col_kernel_gpu<<<_dim_conn, active_col_num, 0, _stream>>>(
		_dim_conn,
		_dim_mcu,
		ptr_ii,
		ptr_ssj,
		ptr_pij,
		ptr_eij,
		ptr_zi2,
		ptr_zj2,
		ptr_tij,
		simstep,
		_taupdt * prn,
		_tauedt,
		_tauzidt,
		_tauzjdt,
		_kftj,
		_norm_frac_bit,
		_p_frac_bit
	);
	CUDA_POST_KERNEL_CHECK;
}

}
}

#endif
