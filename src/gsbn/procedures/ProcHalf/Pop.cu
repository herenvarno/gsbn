#include "hip/hip_runtime.h"
#include "gsbn/procedures/ProcHalf/Pop.hpp"

#ifndef CPU_ONLY

#include "gsbn/Fp16.cuh"

namespace gsbn{
namespace proc_half{

void Pop::update_rnd_gpu(){
	float *ptr_uniform01= _rnd_uniform01->mutable_gpu_data();
	float *ptr_normal= _rnd_normal->mutable_gpu_data();
	int size = _dim_hcu * _dim_mcu;
	_rnd.gen_uniform01_gpu(ptr_uniform01, size);
	_rnd.gen_normal_gpu(ptr_normal, size, 0, _snoise);
}

//#ifndef OLD_CUDA

/*
 * CUDA VERSION > 7.5 supports Half-float datatype
 */
/*
__global__ void update_sup_kernel_gpu(
	int dim_proj,
	int dim_hcu,
	int dim_mcu,
	const half *ptr_epsc,
	const half *ptr_bj,
	const half *ptr_lginp,
	const half *ptr_wmask,
	const float *ptr_rnd_normal,
	const float *ptr_rnd_uniform01,
	half* ptr_dsup,
	half* ptr_act,
	int* ptr_spk,
	float wgain,
	float lgbias,
	float igain,
	float taumdt,
	float wtagain,
	float maxfqdt
){
	extern __shared__ float shmem[];

	int i=blockIdx.x;
	int j=threadIdx.x;
	int idx = i*dim_mcu+j;

	float wsup=0;
	int offset=0;
	int mcu_num_in_pop = dim_hcu * dim_mcu;
	for(int m=0; m<dim_proj; m++){
		wsup += ptr_bj[offset+idx] + ptr_epsc[offset+idx];
		offset += mcu_num_in_pop;
	}

	__shared__ float wmask;
	if(j==0){
		wmask = ptr_wmask[i];
	}
	
	__syncthreads();
	float sup = lgbias + igain * ptr_lginp[idx]+ ptr_rnd_normal[idx];
	sup += (wgain * wmask) * wsup;

	float dsup = ptr_dsup[idx];
	dsup += (sup - dsup)*taumdt;
	ptr_dsup[idx] = dsup;
	
	float* ptr_sh_dsup=&shmem[0];
	ptr_sh_dsup[j] = dsup;
	__syncthreads();
	if(j==0){
		for(int n=1; n<dim_mcu; n++){
			if(ptr_sh_dsup[0]<ptr_sh_dsup[n]){
				ptr_sh_dsup[0] = ptr_sh_dsup[n];
			}
		}
	}
	__syncthreads();
	float maxdsup = ptr_sh_dsup[0];
	float maxact = exp(wtagain*maxdsup);
	float act = exp(wtagain*(dsup-maxdsup));
	if(maxact<1){
		act *= maxact;
	}
	float* ptr_sh_act=&shmem[0];
	__syncthreads();
	ptr_sh_act[j] = act;
	__syncthreads();
	if(j==0){
		for(int n=1; n<dim_mcu; n++){
			ptr_sh_act[0]+=ptr_sh_act[n];
		}
	}
	__syncthreads();
	float vsum = ptr_sh_act[0];
	if(vsum>1){
		act /= vsum;
	}
	ptr_act[idx] = act;
	ptr_spk[idx] = int(ptr_rnd_uniform01[idx]<act*maxfqdt);

}


void Pop::update_sup_gpu(){
	const int *ptr_conf = static_cast<const int*>(_conf->cpu_data());
	int lginp_idx= ptr_conf[Database::IDX_CONF_STIM];
	int wmask_idx= ptr_conf[Database::IDX_CONF_GAIN_MASK]+_hcu_start;
	const half* ptr_wmask = static_cast<const half*>(_wmask->gpu_data(wmask_idx));
	const half* ptr_epsc = static_cast<const half*>(_epsc->gpu_data());
	const half* ptr_bj = static_cast<const half*>(_bj->gpu_data());
	const half* ptr_lginp = static_cast<const half*>(_lginp->gpu_data(lginp_idx))+_mcu_start;
	const float* ptr_rnd_normal = _rnd_normal->gpu_data();
	const float* ptr_rnd_uniform01 = _rnd_uniform01->gpu_data();
	fp16 *ptr_dsup = static_cast<half*>(_dsup->mutable_gpu_data());
	fp16 *ptr_act = static_cast<half*>(_act->mutable_gpu_data());
	int *ptr_spk = _spike->mutable_gpu_data();
	
	update_sup_kernel_gpu<<<_dim_hcu, _dim_mcu, _dim_mcu*sizeof(float), _stream>>>(
		_dim_proj,
		_dim_hcu,
		_dim_mcu,
		ptr_epsc,
		ptr_bj,
		ptr_lginp,
		ptr_wmask,
		ptr_rnd_normal,
		ptr_rnd_uniform01,
		ptr_dsup,
		ptr_act,
		ptr_spk,
		_wgain,
		_lgbias,
		_igain,
		_taumdt,
		_wtagain,
		_maxfqdt
	);
	CUDA_POST_KERNEL_CHECK;
}

*/
//#else
/*
 * CUDA VERSION < 7.5 doesn't support Half-float datatype
 */
__global__ void update_sup_kernel_gpu(
	int dim_proj,
	int dim_hcu,
	int dim_mcu,
	const fp16 *ptr_epsc,
	const fp16 *ptr_bj,
	const float *ptr_lginp,
	const float *ptr_wmask,
	const float *ptr_rnd_normal,
	const float *ptr_rnd_uniform01,
	fp16* ptr_dsup,
	fp16* ptr_act,
	int* ptr_spk,
	float wgain,
	float lgbias,
	float igain,
	float taumdt,
	float wtagain,
	float maxfqdt
){
	extern __shared__ float shmem[];

	int i=blockIdx.x;
	int j=threadIdx.x;
	int idx = i*dim_mcu+j;

	float wsup=0;
	int offset=0;
	int mcu_num_in_pop = dim_hcu * dim_mcu;
	for(int m=0; m<dim_proj; m++){
		wsup += fp16_to_fp32_gpu(ptr_bj[offset+idx]) + fp16_to_fp32_gpu(ptr_epsc[offset+idx]);
		offset += mcu_num_in_pop;
	}

	__shared__ float wmask;
	if(j==0){
		wmask = ptr_wmask[i];
	}
	
	__syncthreads();
	float sup = lgbias + igain * ptr_lginp[idx]+ ptr_rnd_normal[idx];
	sup += (wgain * wmask) * wsup;

	float dsup = fp16_to_fp32_gpu(ptr_dsup[idx]);
	dsup += (sup - dsup)*taumdt;
	ptr_dsup[idx] = fp32_to_fp16_gpu(dsup);
	
	float* ptr_sh_dsup=&shmem[0];
	ptr_sh_dsup[j] = dsup;
	__syncthreads();
	if(j==0){
		for(int n=1; n<dim_mcu; n++){
			if(ptr_sh_dsup[0]<ptr_sh_dsup[n]){
				ptr_sh_dsup[0] = ptr_sh_dsup[n];
			}
		}
	}
	__syncthreads();
	float maxdsup = ptr_sh_dsup[0];
	float maxact = exp(wtagain*maxdsup);
	float act = exp(wtagain*(dsup-maxdsup));
	if(maxact<1){
		act *= maxact;
	}
	float* ptr_sh_act=&shmem[0];
	__syncthreads();
	ptr_sh_act[j] = act;
	__syncthreads();
	if(j==0){
		for(int n=1; n<dim_mcu; n++){
			ptr_sh_act[0]+=ptr_sh_act[n];
		}
	}
	__syncthreads();
	float vsum = ptr_sh_act[0];
	if(vsum>1){
		act /= vsum;
	}
	ptr_act[idx] = fp32_to_fp16_gpu(act);
	ptr_spk[idx] = int(ptr_rnd_uniform01[idx]<act*maxfqdt);

}


void Pop::update_sup_gpu(){
	const int *ptr_conf = static_cast<const int*>(_conf->cpu_data());
	int lginp_idx= ptr_conf[Database::IDX_CONF_STIM];
	int wmask_idx= ptr_conf[Database::IDX_CONF_GAIN_MASK]+_hcu_start;
	const float* ptr_wmask = _wmask->gpu_data(wmask_idx);
	const fp16* ptr_epsc = _epsc->gpu_data();
	const fp16* ptr_bj = _bj->gpu_data();
	const float* ptr_lginp = _lginp->gpu_data(lginp_idx)+_mcu_start;
	const float* ptr_rnd_normal = _rnd_normal->gpu_data();
	const float* ptr_rnd_uniform01 = _rnd_uniform01->gpu_data();
	fp16 *ptr_dsup = _dsup->mutable_gpu_data();
	fp16 *ptr_act = _act->mutable_gpu_data();
	int *ptr_spk = _spike->mutable_gpu_data();
	
	update_sup_kernel_gpu<<<_dim_hcu, _dim_mcu, _dim_mcu*sizeof(float), _stream>>>(
		_dim_proj,
		_dim_hcu,
		_dim_mcu,
		ptr_epsc,
		ptr_bj,
		ptr_lginp,
		ptr_wmask,
		ptr_rnd_normal,
		ptr_rnd_uniform01,
		ptr_dsup,
		ptr_act,
		ptr_spk,
		_wgain,
		_lgbias,
		_igain,
		_taumdt,
		_wtagain,
		_maxfqdt
	);
	CUDA_POST_KERNEL_CHECK;
}

//#endif

}
}

#endif
