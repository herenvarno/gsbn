#include "hip/hip_runtime.h"
#include "gsbn/procedures/ProcUpdFpP32O32/Proj.hpp"

#ifndef CPU_ONLY

namespace gsbn{
namespace proc_upd_fp_p32_o32{

__global__ void update_full_kernel_gpu(
	int dim_conn,
	int dim_mcu,
	float *ptr_pi,
	float *ptr_ei,
	float *ptr_zi,
	int *ptr_ti,
	const float *ptr_pj,
	float *ptr_pij,
	float *ptr_eij,
	float *ptr_zi2,
	float *ptr_zj2,
	int *ptr_tij,
	float *ptr_wij,
	int simstep,
	float kp,
	float ke,
	float kzi,
	float kzj,
	float wgain,
	float eps,
	float eps2
){
	int i=blockIdx.y*gridDim.x+blockIdx.x;
	int j=threadIdx.x;
	
	__shared__ float sh_pi;
	if(j==0){
		float pi = ptr_pi[i];
		float zi = ptr_zi[i];
		int ti = ptr_ti[i];
		int pdt = simstep - ti;
		if(pdt<=0){
			ptr_ti[i]=simstep;
		}else{
			float ei = ptr_ei[i];
			pi = (pi - ((ei*kp*kzi - ei*ke*kp + ke*kp*zi)/(ke - kp) +
				(ke*kp*zi)/(kp - kzi))/(ke - kzi))/exp(kp*pdt) +
				((exp(kp*pdt - ke*pdt)*(ei*kp*kzi - ei*ke*kp + ke*kp*zi))/(ke - kp) +
				(ke*kp*zi*exp(kp*pdt - kzi*pdt))/(kp - kzi))/(exp(kp*pdt)*(ke - kzi));
			ei = (ei - (ke*zi)/(ke - kzi))/exp(ke*pdt) +
				(ke*zi*exp(ke*pdt - kzi*pdt))/(exp(ke*pdt)*(ke - kzi));
			zi = zi*exp(-kzi*pdt);
			ti = simstep;
		
			ptr_pi[i] = pi;
			ptr_ei[i] = ei;
			ptr_zi[i] = zi;
			ptr_ti[i] = ti;
		}
		sh_pi = pi;
	}
	__syncthreads();
	
	int index = i*dim_mcu+j;
	
	int tij = ptr_tij[index];
	float zi2 = ptr_zi2[index];
	int pdt = simstep - tij;
	if(pdt<=0){
		ptr_tij[index]=simstep;
	}else{
		float pij = ptr_pij[index];
		float eij = ptr_eij[index];
		float zj2 = ptr_zj2[index];
	
		pij = (pij + ((eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2)/(ke - kp) -
			(ke*kp*zi2*zj2)/(kzi - kp + kzj))/(kzi - ke + kzj))/exp(kp*pdt) -
			((exp(kp*pdt - ke*pdt)*(eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2))/(ke - kp) -
			(ke*kp*zi2*zj2*exp(kp*pdt - kzi*pdt - kzj*pdt))/
			(kzi - kp + kzj))/(exp(kp*pdt)*(kzi - ke + kzj));
		eij = (eij + (ke*zi2*zj2)/(kzi - ke + kzj))/exp(ke*pdt) -
			(ke*zi2*zj2)/(exp(kzi*pdt)*exp(kzj*pdt)*(kzi - ke + kzj));
		zi2 = zi2*exp(-kzi*pdt);
		zj2 = zj2*exp(-kzj*pdt);
		tij = simstep;
			 	
		ptr_pij[index] = pij;
		ptr_eij[index] = eij;
		ptr_zi2[index] = zi2;
		ptr_zj2[index] = zj2;
		ptr_tij[index] = tij;
			
		// update wij and epsc
		float wij;
		if(kp){
			float pi = sh_pi;
			float pj = ptr_pj[i/dim_conn*dim_mcu + j];
			wij = wgain * log((pij + eps2)/((pi + eps)*(pj + eps)));
			ptr_wij[index] = wij;
		}
	}
}

__global__ void update_j_kernel_gpu(
	int n,
	const int8_t *ptr_sj,
	float *ptr_pj,
	float *ptr_ej,
	float *ptr_zj,
	float *ptr_bj,
	float *ptr_epsc,
	float kp,
	float ke,
	float kzj,
	float kzi,
	float kftj,
	float bgain,
	float eps
){
	CUDA_KERNEL_LOOP(idx, n){
		float pj = ptr_pj[idx];
		float ej = ptr_ej[idx];
		float zj = ptr_zj[idx];
		int sj = ptr_sj[idx];
		
		ptr_epsc[idx] *= (1-kzi);

		if(kp){
			float bj = bgain * log(pj + eps);
			ptr_bj[idx]=bj;
		}

		pj += (ej - pj)*kp;
		ej += (zj - ej)*ke;
		zj *= (1-kzj);
		if(sj>0){
			zj += kftj;
		}
	
		ptr_pj[idx] = pj;
		ptr_ej[idx] = ej;
		ptr_zj[idx] = zj;
	}
}

__global__ void update_row_kernel_gpu(
	int dim_conn,
	int dim_mcu,
	const int *ptr_ssi,
	float *ptr_pi,
	float *ptr_ei,
	float *ptr_zi,
	int *ptr_ti,
	const float *ptr_pj,
	float *ptr_pij,
	float *ptr_eij,
	float *ptr_zi2,
	float *ptr_zj2,
	int *ptr_tij,
	float* ptr_wij,
	float* ptr_epsc,
	int simstep,
	float kp,
	float ke,
	float kzi,
	float kzj,
	float kfti,
	float wgain,
	float eps,
	float eps2
){
	int i = blockIdx.x;
	int j = threadIdx.x;
	int row = ptr_ssi[i];
	int col = j;
	int index = row*dim_mcu+col;
	
	__shared__ float sh_pi;
	
	if(j==0){
		float pi = ptr_pi[row];
		float zi = ptr_zi[row];
		int ti = ptr_ti[row];
		int pdt = simstep - ti;
		if(pdt<=0){
			ptr_zi[row] += kfti;
			ptr_ti[row] = simstep;
		}else{
			float ei = ptr_ei[row];
		
			pi = (pi - ((ei*kp*kzi - ei*ke*kp + ke*kp*zi)/(ke - kp) +
				(ke*kp*zi)/(kp - kzi))/(ke - kzi))/exp(kp*pdt) +
				((exp(kp*pdt - ke*pdt)*(ei*kp*kzi - ei*ke*kp + ke*kp*zi))/(ke - kp) +
				(ke*kp*zi*exp(kp*pdt - kzi*pdt))/(kp - kzi))/(exp(kp*pdt)*(ke - kzi));
			ei = (ei - (ke*zi)/(ke - kzi))/exp(ke*pdt) +
				(ke*zi*exp(ke*pdt - kzi*pdt))/(exp(ke*pdt)*(ke - kzi));
			zi = zi*exp(-kzi*pdt) + kfti;
			ti = simstep;
			ptr_pi[row] = pi;
			ptr_ei[row] = ei;
			ptr_zi[row] = zi;
			ptr_ti[row] = ti;
		}
		sh_pi = pi;
	}
	
	__syncthreads();
	
	float pij = ptr_pij[index];
	int tij = ptr_tij[index];
	float zi2 = ptr_zi2[index];
	int pdt = simstep - tij;
	if(pdt<=0){
		ptr_zi2[index] += kfti;
		ptr_tij[index] = simstep;
	}else{
		float eij = ptr_eij[index];
		float zj2 = ptr_zj2[index];
	
		pij = (pij + ((eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2)/(ke - kp) -
			(ke*kp*zi2*zj2)/(kzi - kp + kzj))/(kzi - ke + kzj))/exp(kp*pdt) -
			((exp(kp*pdt - ke*pdt)*(eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2))/(ke - kp) -
			(ke*kp*zi2*zj2*exp(kp*pdt - kzi*pdt - kzj*pdt))/
			(kzi - kp + kzj))/(exp(kp*pdt)*(kzi - ke + kzj));
		eij = (eij + (ke*zi2*zj2)/(kzi - ke + kzj))/exp(ke*pdt) -
			(ke*zi2*zj2)/(exp(kzi*pdt)*exp(kzj*pdt)*(kzi - ke + kzj));
		zi2 = zi2*exp(-kzi*pdt)+kfti;
		zj2 = zj2*exp(-kzj*pdt);
		tij = simstep;
			 	
		ptr_pij[index] = pij;
		ptr_eij[index] = eij;
		ptr_zi2[index] = zi2;
		ptr_zj2[index] = zj2;
		ptr_tij[index] = tij;
		
		float wij;
		int idx_hcu = row / dim_conn;
		int idx_mcu = idx_hcu * dim_mcu + j;
		if(kp){
			float pi = sh_pi;
			float pj = ptr_pj[idx_mcu];
			wij = wgain * log((pij + eps2)/((pi + eps)*(pj + eps)));
			ptr_wij[index] = wij;
		}else{
			wij = ptr_wij[index];
		}
		atomicAdd(&ptr_epsc[idx_mcu], wij);
	}
}

__global__ void update_col_kernel_gpu(
	int dim_conn,
	int dim_mcu,
	const int *ptr_ii,
	const int *ptr_ssj,
	float *ptr_pij,
	float *ptr_eij,
	float *ptr_zi2,
	float *ptr_zj2,
	int *ptr_tij,
	int simstep,
	float kp,
	float ke,
	float kzi,
	float kzj,
	float kftj
){

	int i = blockIdx.x;
	int j = threadIdx.x;

	int row = ptr_ssj[j]/dim_mcu*dim_conn+i;
	if(ptr_ii[row]<0){
		return;
	}
	int col = ptr_ssj[j]%dim_mcu;
	int index = row*dim_mcu+col;
	
	int tij = ptr_tij[index];
	float zj2 = ptr_zj2[index];
	int pdt = simstep - tij;
	if(pdt<=0){
		zj2 += kftj;
		ptr_zj2[index]=zj2;
		ptr_tij[index]=simstep;
	}else{
		float pij = ptr_pij[index];
		float eij = ptr_eij[index];
		float zi2 = ptr_zi2[index];
	
		pij = (pij + ((eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2)/(ke - kp) -
			(ke*kp*zi2*zj2)/(kzi - kp + kzj))/(kzi - ke + kzj))/exp(kp*pdt) -
			((exp(kp*pdt - ke*pdt)*(eij*kp*kzi - eij*ke*kp + eij*kp*kzj + ke*kp*zi2*zj2))/(ke - kp) -
			(ke*kp*zi2*zj2*exp(kp*pdt - kzi*pdt - kzj*pdt))/
			(kzi - kp + kzj))/(exp(kp*pdt)*(kzi - ke + kzj));
		eij = (eij + (ke*zi2*zj2)/(kzi - ke + kzj))/exp(ke*pdt) -
			(ke*zi2*zj2)/(exp(kzi*pdt)*exp(kzj*pdt)*(kzi - ke + kzj));
		zi2 = zi2*exp(-kzi*pdt);
		zj2 = zj2*exp(-kzj*pdt)+kftj;
		tij = simstep;
			 	
		ptr_pij[index] = pij;
		ptr_eij[index] = eij;
		ptr_zi2[index] = zi2;
		ptr_zj2[index] = zj2;
		ptr_tij[index] = tij;
	}
}


void Proj::update_full_gpu(){
	const int *ptr_conf0 = static_cast<const int*>(_conf->cpu_data());
	const float *ptr_conf1 = static_cast<const float*>(_conf->cpu_data());
	int simstep = ptr_conf0[Database::IDX_CONF_TIMESTAMP];
	float prn = ptr_conf1[Database::IDX_CONF_PRN];
	float old_prn = ptr_conf1[Database::IDX_CONF_OLD_PRN];
	if(old_prn!=prn){
		float *ptr_pi = _pi->mutable_gpu_data();
		float *ptr_ei = _ei->mutable_gpu_data();
		float *ptr_zi = _zi->mutable_gpu_data();
		int *ptr_ti = _ti->mutable_gpu_data();
		const float *ptr_pj = _pj->gpu_data();
		float *ptr_pij = _pij->mutable_gpu_data();
		float *ptr_eij = _eij->mutable_gpu_data();
		float *ptr_zi2 = _zi2->mutable_gpu_data();
		float *ptr_zj2 = _zj2->mutable_gpu_data();
		int *ptr_tij = _tij->mutable_gpu_data();
		float *ptr_wij = _wij->mutable_gpu_data();
		const dim3 GRID_SIZE(_dim_conn, _dim_hcu);
		update_full_kernel_gpu<<<GRID_SIZE, _dim_mcu, 0, _stream>>>(
			_dim_conn,
			_dim_mcu,
			ptr_pi,
			ptr_ei,
			ptr_zi,
			ptr_ti,
			ptr_pj,
			ptr_pij,
			ptr_eij,
			ptr_zi2,
			ptr_zj2,
			ptr_tij,
			ptr_wij,
			simstep,
			_taupdt*old_prn,
			_tauedt,
			_tauzidt,
			_tauzjdt,
			_wgain,
			_eps,
			_eps2
		);
		CUDA_POST_KERNEL_CHECK;
	}
}

void Proj::update_j_gpu(){
	const float *ptr_conf = static_cast<const float*>(_conf->cpu_data());
	float prn = ptr_conf[Database::IDX_CONF_PRN];
	float *ptr_pj = _pj->mutable_gpu_data();
	float *ptr_ej = _ej->mutable_gpu_data();
	float *ptr_zj = _zj->mutable_gpu_data();
	float *ptr_bj = _bj->mutable_gpu_data()+_proj_in_pop*_dim_hcu*_dim_mcu;
	float *ptr_epsc = _epsc->mutable_gpu_data()+_proj_in_pop*_dim_hcu*_dim_mcu;
	const int8_t *ptr_sj = _sj->gpu_data();

	update_j_kernel_gpu<<<GSBN_GET_BLOCKS(_dim_hcu*_dim_mcu), GSBN_GET_THREADS(_dim_hcu*_dim_mcu), 0, _stream>>>(
		_dim_hcu*_dim_mcu,
		ptr_sj,
		ptr_pj,
		ptr_ej,
		ptr_zj,
		ptr_bj,
		ptr_epsc,
		_taupdt*prn,
		_tauedt,
		_tauzjdt,
		_tauzidt,
		_kftj,
		_bgain,
		_eps
	);
	CUDA_POST_KERNEL_CHECK;
}

void Proj::update_ss_gpu(){
	// get active in spike
	CONST_HOST_VECTOR(int8_t, *v_si) = _si->cpu_vector();
	CONST_HOST_VECTOR(int, *v_ii) = _ii->cpu_vector();
	CONST_HOST_VECTOR(int, *v_di) = _di->cpu_vector();
	HOST_VECTOR(int, *v_qi) = _qi->mutable_cpu_vector();
	HOST_VECTOR(int, *v_ssi) = _ssi->mutable_cpu_vector();
	
	v_ssi->clear();
	for(int i=0; i<_dim_conn * _dim_hcu; i++){
		if((*v_ii)[i]<0){
			continue;
		}
		(*v_qi)[i] >>= 1;
		if((*v_qi)[i] & 0x01){
			v_ssi->push_back(i);
		}
	
		int spk = (*v_si)[(*v_ii)[i]];
		if(spk>0){
			(*v_qi)[i] |= (0x01 << (*v_di)[i]);
		}
	}
	
	// get active out spike
	CONST_HOST_VECTOR(int8_t, *v_sj) = _sj->cpu_vector();
	HOST_VECTOR(int, *v_ssj) = _ssj->mutable_cpu_vector();
	v_ssj->clear();
	for(int i=0; i<v_sj->size(); i++){
		if((*v_sj)[i]>0){
			v_ssj->push_back(i);
		}
	}
}

void Proj::update_row_gpu(){
	int active_row_num = _ssi->gpu_vector()->size();
	if(active_row_num<=0){
		return;
	}

	const int *ptr_conf0 = static_cast<const int*>(_conf->cpu_data());
	const float *ptr_conf1 = static_cast<const float*>(_conf->cpu_data());
	int simstep = ptr_conf0[Database::IDX_CONF_TIMESTAMP];
	float prn = ptr_conf1[Database::IDX_CONF_PRN];
	
	float *ptr_pi = _pi->mutable_gpu_data();
	float *ptr_ei = _ei->mutable_gpu_data();
	float *ptr_zi = _zi->mutable_gpu_data();
	int *ptr_ti = _ti->mutable_gpu_data();
	const float *ptr_pj = _pj->gpu_data();
	float *ptr_pij = _pij->mutable_gpu_data();
	float *ptr_eij = _eij->mutable_gpu_data();
	float *ptr_zi2 = _zi2->mutable_gpu_data();
	float *ptr_zj2 = _zj2->mutable_gpu_data();
	int *ptr_tij = _tij->mutable_gpu_data();
	float *ptr_wij = _wij->mutable_gpu_data();
	float *ptr_epsc = _epsc->mutable_gpu_data()+ _proj_in_pop * _dim_hcu * _dim_mcu;
	
	const int *ptr_ssi = _ssi->gpu_data();

	update_row_kernel_gpu<<<active_row_num, _dim_mcu, 0, _stream>>>(
		_dim_conn,
		_dim_mcu,
		ptr_ssi,
		ptr_pi,
		ptr_ei,
		ptr_zi,
		ptr_ti,
		ptr_pj,
		ptr_pij,
		ptr_eij,
		ptr_zi2,
		ptr_zj2,
		ptr_tij,
		ptr_wij,
		ptr_epsc,
		simstep,
		_taupdt*prn,
		_tauedt,
		_tauzidt,
		_tauzjdt,
		_kfti,
		_wgain,
		_eps,
		_eps2
	);
}

void Proj::update_col_gpu(){
	int active_col_num = _ssj->gpu_vector()->size();
	if(active_col_num<=0){
		return;
	}
	const int *ptr_conf0 = static_cast<const int*>(_conf->cpu_data());
	const float *ptr_conf1 = static_cast<const float*>(_conf->cpu_data());
	int simstep = ptr_conf0[Database::IDX_CONF_TIMESTAMP];
	float prn = ptr_conf1[Database::IDX_CONF_PRN];
	
	float *ptr_pij = _pij->mutable_gpu_data();
	float *ptr_eij = _eij->mutable_gpu_data();
	float *ptr_zi2 = _zi2->mutable_gpu_data();
	float *ptr_zj2 = _zj2->mutable_gpu_data();
	int *ptr_tij = _tij->mutable_gpu_data();
	
	const int *ptr_ii = _ii->gpu_data();
	const int *ptr_ssj = _ssj->gpu_data();
	
	update_col_kernel_gpu<<<_dim_conn, active_col_num, 0, _stream>>>(
		_dim_conn,
		_dim_mcu,
		ptr_ii,
		ptr_ssj,
		ptr_pij,
		ptr_eij,
		ptr_zi2,
		ptr_zj2,
		ptr_tij,
		simstep,
		_taupdt * prn,
		_tauedt,
		_tauzidt,
		_tauzjdt,
		_kftj
	);
}

}
}

#endif
