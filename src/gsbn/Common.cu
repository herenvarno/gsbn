#include "gsbn/Common.hpp"

namespace gsbn{

#ifndef CPU_ONLY

static hipblasHandle_t _cublas_handle;

hipblasHandle_t cublas_handle(){
	return _cublas_handle;
}

void common_init(){
CUBLAS_CHECK(hipblasCreate(&_cublas_handle));
}

#endif

}
