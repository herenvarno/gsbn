#include "hip/hip_runtime.h"
#include "gsbn/procedures/ProcNetBatch/Pop.hpp"

#ifndef CPU_ONLY

namspace gsbn{
namespace proc_net_batch{

void Pop::update_rnd_gpu(){
	float *ptr_uniform01= _rnd_uniform01->mutable_gpu_data();
	float *ptr_normal= _rnd_normal->mutable_gpu_data();
	int size = _dim_hcu * _dim_mcu;
	_rnd.gen_uniform01_gpu(ptr_uniform01, size);
	_rnd.gen_normal_gpu(ptr_normal, size, 0, _snoise);
}


__global__ void update_sup_kernel_gpu(
	int dim_proj,
	int dim_hcu,
	int dim_mcu,
	const float *ptr_epsc,
	const float *ptr_bj,
	const float *ptr_lginp,
	const float *ptr_wmask,
	const float *ptr_rnd_normal,
	const float *ptr_rnd_uniform01,
	float* ptr_dsup,
	float* ptr_act,
	int* ptr_spk,
	float wgain,
	float lgbias,
	float igain,
	float taumdt,
	float wtagain,
	float maxfqdt
){
	extern __shared__ float shmem[];

	int i=blockIdx.x;
	int j=threadIdx.x;
	int idx = i*dim_mcu+j;

	float wsup=0;
	int offset=0;
	int mcu_num_in_pop = dim_hcu * dim_mcu;
	for(int m=0; m<dim_proj; m++){
		wsup += ptr_bj[offset+idx] + ptr_epsc[offset+idx];
		offset += mcu_num_in_pop;
	}

	__shared__ float wmask;
	if(j==0){
		wmask = ptr_wmask[i];
	}
	
	__syncthreads();
	float sup = lgbias + igain * ptr_lginp[idx]+ ptr_rnd_normal[idx];
	sup += (wgain * wmask) * wsup;

	float dsup = ptr_dsup[idx];
	dsup += (sup - dsup)*taumdt;
	ptr_dsup[idx] = dsup;
	
	float* ptr_sh_dsup=&shmem[0];
	ptr_sh_dsup[j] = dsup;
	__syncthreads();
	if(j==0){
		for(int n=1; n<dim_mcu; n++){
			if(ptr_sh_dsup[0]<ptr_sh_dsup[n]){
				ptr_sh_dsup[0] = ptr_sh_dsup[n];
			}
		}
	}
	__syncthreads();
	float maxdsup = ptr_sh_dsup[0];
	float maxact = exp(wtagain*maxdsup);
	float act = exp(wtagain*(dsup-maxdsup));
	if(maxact<1){
		act *= maxact;
	}
	float* ptr_sh_act=&shmem[0];
	__syncthreads();
	ptr_sh_act[j] = act;
	__syncthreads();
	if(j==0){
		for(int n=1; n<dim_mcu; n++){
			ptr_sh_act[0]+=ptr_sh_act[n];
		}
	}
	__syncthreads();
	float vsum = ptr_sh_act[0];
	if(vsum>1){
		act /= vsum;
	}
	ptr_act[idx] = act;
	ptr_spk[idx] = int(ptr_rnd_uniform01[idx]<act*maxfqdt);

}

void Pop::update_sup_gpu(){
	const int *ptr_conf = static_cast<const int*>(_conf->cpu_data());
	int lginp_idx= ptr_conf[Database::IDX_CONF_STIM];
	int wmask_idx= ptr_conf[Database::IDX_CONF_GAIN_MASK]+_hcu_start;
	const float* ptr_wmask = _wmask->gpu_data(wmask_idx);
	const float* ptr_epsc = _epsc->gpu_data();
	const float* ptr_bj = _bj->gpu_data();
	const float *ptr_lginp = _lginp->gpu_data(lginp_idx)+_mcu_start;
	const float *ptr_rnd_normal = _rnd_normal->gpu_data();
	const float *ptr_rnd_uniform01 = _rnd_uniform01->gpu_data();
	float *ptr_dsup = _dsup->mutable_gpu_data();
	float *ptr_act = _act->mutable_gpu_data();
	int *ptr_spk = _spike->mutable_gpu_data()+_mcu_start;
	
	update_kernel_gpu<<<dim_hcu, dim_mcu, dim_mcu*sizeof(float), _stream>>>(
		dim_proj,
		dim_hcu,
		dim_mcu,
		ptr_epsc,
		ptr_bj,
		ptr_lginp,
		ptr_wmask,
		ptr_rnd_normal,
		ptr_rnd_uniform01,
		ptr_dsup,
		ptr_act,
		ptr_spk,
		_wgain,
		_lgbias,
		_igain,
		_taumdt,
		_wtagain,
		_maxfqdt
	);
	CUDA_POST_KERNEL_CHECK;
}


}
}

#endif
